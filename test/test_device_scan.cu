#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of DeviceScan utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <hipcub/hipcub.hpp>
#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose           = false;
int                     g_timing_iterations = 0;
int                     g_repeat            = 0;
CachingDeviceAllocator  g_allocator;


//---------------------------------------------------------------------
// Dispatch to different DeviceScan entrypoints
//---------------------------------------------------------------------


/**
 * Exclusive scan specialization
 */
template <
    typename InputIteratorRA,
    typename OutputIteratorRA,
    typename ScanOp,
    typename Identity,
    typename SizeT>
__host__ __device__ __forceinline__
hipError_t Dispatch(
    void                        *d_temp_storage,                    ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
    size_t                      &temp_storage_bytes,                ///< [in,out] Size in bytes of \t d_temp_storage allocation.
    InputIteratorRA             d_in,
    OutputIteratorRA            d_out,
    ScanOp                      scan_op,
    Identity                    identity,
    SizeT                       num_items,
    hipStream_t                stream              = 0,
    bool                        stream_synchronous  = false)
{
    return DeviceScan::ExclusiveScan(d_temp_storage, temp_storage_bytes, d_in, d_out, scan_op, identity, num_items, stream, stream_synchronous);
}


/**
 * Exclusive sum specialization
 */
template <
    typename InputIteratorRA,
    typename OutputIteratorRA,
    typename T,
    typename Identity,
    typename SizeT>
__host__ __device__ __forceinline__
hipError_t Dispatch(
    void                        *d_temp_storage,                    ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
    size_t                      &temp_storage_bytes,                ///< [in,out] Size in bytes of \t d_temp_storage allocation.
    InputIteratorRA             d_in,
    OutputIteratorRA            d_out,
    Sum                         scan_op,
    Identity                    identity,
    SizeT                       num_items,
    hipStream_t                stream              = 0,
    bool                        stream_synchronous  = false)
{
    return DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, stream, stream_synchronous);
}


/**
 * Inclusive scan specialization
 */
template <
    typename InputIteratorRA,
    typename OutputIteratorRA,
    typename ScanOp,
    typename SizeT>
__host__ __device__ __forceinline__
hipError_t Dispatch(
    void                        *d_temp_storage,                    ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
    size_t                      &temp_storage_bytes,                ///< [in,out] Size in bytes of \t d_temp_storage allocation.
    InputIteratorRA             d_in,
    OutputIteratorRA            d_out,
    ScanOp                      scan_op,
    NullType                    identity,
    SizeT                       num_items,
    hipStream_t                stream              = 0,
    bool                        stream_synchronous  = false)
{
    return DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_in, d_out, scan_op, num_items, stream, stream_synchronous);
}


/**
 * Inclusive sum specialization
 */
template <
    typename InputIteratorRA,
    typename OutputIteratorRA,
    typename T,
    typename SizeT>
__host__ __device__ __forceinline__
hipError_t Dispatch(
    void                        *d_temp_storage,                    ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
    size_t                      &temp_storage_bytes,                ///< [in,out] Size in bytes of \t d_temp_storage allocation.
    InputIteratorRA             d_in,
    OutputIteratorRA            d_out,
    Sum                         scan_op,
    NullType                    identity,
    SizeT                       num_items,
    hipStream_t                stream              = 0,
    bool                        stream_synchronous  = false)
{
    return DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, stream, stream_synchronous);
}

//---------------------------------------------------------------------
// CUDA Nested Parallelism Test Kernel
//---------------------------------------------------------------------

/**
 * Simple wrapper kernel to invoke DeviceScan
 */
/*
template <
    bool                STREAM_SYNCHRONOUS,
    typename            InputIteratorRA,
    typename            OutputIteratorRA,
    typename            ScanOp>
__global__ void CnpScan(
    InputIteratorRA     d_in,
    OutputIteratorRA    d_out,
    int                 num_items,
    ScanOp              scan_op,
    int                 iterations,
    hipError_t*        d_cnp_error)
{
    hipError_t error = hipSuccess;

#ifdef CUB_RUNTIME_ENABLED
    for (int i = 0; i < iterations; ++i)
    {
        error = DeviceScan::Scan(d_in, d_out, num_items, scan_op, 0, STREAM_SYNCHRONOUS);
    }
#else
    error = hipErrorNotSupported;
#endif

    *d_cnp_error = error;
}
*/

//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize exclusive-scan problem (and solution)
 */
template <
    typename     T,
    typename     ScanOp,
    typename     IdentityT>
T Initialize(
    GenMode      gen_mode,
    T            *h_in,
    T            *h_reference,
    int          num_items,
    ScanOp       scan_op,
    IdentityT    identity)
{
    T inclusive = identity;
    T aggregate = identity;

    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
        h_reference[i] = inclusive;
        inclusive = scan_op(inclusive, h_in[i]);
        aggregate = scan_op(aggregate, h_in[i]);
    }

    return aggregate;
}


/**
 * Initialize inclusive-scan problem (and solution)
 */
template <
    typename     T,
    typename     ScanOp>
T Initialize(
    GenMode      gen_mode,
    T            *h_in,
    T            *h_reference,
    int          num_items,
    ScanOp       scan_op,
    NullType)
{
    T inclusive;
    T aggregate;
    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
        if (i == 0)
        {
            inclusive = h_in[0];
            aggregate = h_in[0];
        }
        else
        {
            inclusive = scan_op(inclusive, h_in[i]);
            aggregate = scan_op(aggregate, h_in[i]);
        }
        h_reference[i] = inclusive;
    }

    return aggregate;
}




//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

/**
 * Test DeviceScan
 */
template <
    typename        T,
    typename        ScanOp,
    typename        IdentityT>
void Test(
    int             num_items,
    GenMode         gen_mode,
    ScanOp          scan_op,
    IdentityT       identity,
    char*           type_string)
{
    int compare = 0;
    int cnp_compare = 0;

    printf("%s hipcub::DeviceScan::%s %d items, %s %d-byte elements, gen-mode %s\n",
        (Equals<IdentityT, NullType>::VALUE) ? "Inclusive" : "Exclusive",
        (Equals<ScanOp, Sum>::VALUE) ? "Sum" : "Scan",
        num_items,
        type_string,
        (int) sizeof(T),
        (gen_mode == RANDOM) ? "RANDOM" : (gen_mode == SEQ_INC) ? "SEQUENTIAL" : "HOMOGENOUS");
    fflush(stdout);

    // Allocate host arrays
    T*  h_in = new T[num_items];
    T*  h_reference = new T[num_items];

    // Initialize problem
    Initialize(gen_mode, h_in, h_reference, num_items, scan_op, identity);

    // Allocate device arrays
    T*              d_in = NULL;
    T*              d_out = NULL;
    hipError_t*    d_cnp_error = NULL;
    void            *d_temporary_storage = NULL;
    size_t          temporary_storage_bytes = 0;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in,          sizeof(T) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out,         sizeof(T) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_cnp_error,   sizeof(hipError_t) * 1));

    // Initialize device arrays
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_items, hipMemcpyHostToDevice));
    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * num_items));

    // Allocate temporary storage
    CubDebugExit(Dispatch(d_temporary_storage, temporary_storage_bytes, d_in, d_out, scan_op, identity, num_items, 0, true));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temporary_storage, temporary_storage_bytes));

    // Run warmup/correctness iteration
    printf("Host dispatch:\n"); fflush(stdout);
    CubDebugExit(Dispatch(d_temporary_storage, temporary_storage_bytes, d_in, d_out, scan_op, identity, num_items, 0, true));

    // Check for correctness (and display results, if specified)
    compare = CompareDeviceResults(h_reference, d_out, num_items, true, g_verbose);
    printf("\t%s", compare ? "FAIL" : "PASS");

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Performance
    GpuTimer gpu_timer;
    float elapsed_millis = 0.0;
    for (int i = 0; i < g_timing_iterations; i++)
    {
        gpu_timer.Start();

        CubDebugExit(Dispatch(d_temporary_storage, temporary_storage_bytes, d_in, d_out, scan_op, identity, num_items));

        gpu_timer.Stop();
        elapsed_millis += gpu_timer.ElapsedMillis();
    }
    if (g_timing_iterations > 0)
    {
        float avg_millis = elapsed_millis / g_timing_iterations;
        float grate = float(num_items) / avg_millis / 1000.0 / 1000.0;
        float gbandwidth = grate * sizeof(T) * 2;
        printf(", %.3f avg ms, %.3f billion items/s, %.3f GB/s", avg_millis, grate, gbandwidth);
    }

    printf("\n\n");


    // Evaluate using CUDA nested parallelism
/*
#if (CUB_CNP == 1)

    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * 1));

    // Run warmup/correctness iteration
    printf("\nDevice dispatch:\n"); fflush(stdout);
    CnpScan<true><<<1,1>>>(d_in, d_out, num_items, scan_op, 1, d_cnp_error);

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Check if we were compiled and linked for CNP
    hipError_t h_cnp_error;
    CubDebugExit(hipMemcpy(&h_cnp_error, d_cnp_error, sizeof(hipError_t) * 1, hipMemcpyDeviceToHost));
    if (h_cnp_error == hipErrorInvalidConfiguration)
    {
        printf("CNP not supported");
    }
    else
    {
        CubDebugExit(h_cnp_error);

        // Check for correctness (and display results, if specified)
        cnp_compare = CompareDeviceResults(h_reference, d_out, 1, g_verbose, g_verbose);
        printf("\n%s", cnp_compare ? "FAIL" : "PASS");

        // Performance
        gpu_timer.Start();

        CnpScan<false><<<1,1>>>(d_in, d_out, num_items, scan_op, g_timing_iterations, d_cnp_error);

        gpu_timer.Stop();
        elapsed_millis = gpu_timer.ElapsedMillis();

        if (g_timing_iterations > 0)
        {
            float avg_millis = elapsed_millis / g_timing_iterations;
            float grate = float(num_items) / avg_millis / 1000.0 / 1000.0;
            float gbandwidth = grate * sizeof(T) * 2;
            printf(", %.3f avg ms, %.3f billion items/s, %.3f GB/s\n", avg_millis, grate, gbandwidth);
        }
        else
        {
            printf("\n");
        }
    }

#endif
*/

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_cnp_error) CubDebugExit(g_allocator.DeviceFree(d_cnp_error));
    if (d_temporary_storage) CubDebugExit(g_allocator.DeviceFree(d_temporary_storage));

    // Correctness asserts
    AssertEquals(0, compare);
    AssertEquals(0, cnp_compare);
}


/**
 * Iterative different gen modes
 */
template <
    typename        T,
    typename        ScanOp,
    typename        Identity>
void Test(
    int             num_items,
    ScanOp          scan_op,
    Identity        identity,
    char*           type_string)
{
    Test<T>(num_items, UNIFORM, scan_op, identity, type_string);
    Test<T>(num_items, RANDOM, scan_op, identity, type_string);
}


/**
 * Iterate inclusive/exclusive
 */
template <
    typename        T,
    typename        ScanOp>
void Test(
    int             num_items,
    ScanOp          scan_op,
    char*           type_string)
{
    Test<T>(num_items, scan_op, T(), type_string);          // exclusive
    Test<T>(num_items, scan_op, NullType(), type_string);   // inclusive
}


/**
 * Iterate sum/scan
 */
template <
    typename        T>
void TestOp(
    int             num_items,
    char*           type_string)
{
    Test<T>(num_items, Sum(), type_string);
    Test<T>(num_items, Max(), type_string);
}


/**
 * Iterate input sizes
 */
template <
    typename        T>
void Test(
    int             num_items,
    char*           type_string)
{
    if (num_items < 0)
    {
        TestOp<T>(1,        type_string);
        TestOp<T>(100,      type_string);
        TestOp<T>(10000,    type_string);
        TestOp<T>(1000000,  type_string);
    }
    else
    {
        TestOp<T>(num_items, type_string);
    }
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_items = -1;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    bool quick = args.CheckCmdLineFlag("quick");
    args.GetCmdLineArgument("n", num_items);
    args.GetCmdLineArgument("i", g_timing_iterations);
    args.GetCmdLineArgument("repeat", g_repeat);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<input items> "
            "[--i=<timing iterations> "
            "[--device=<device-id>] "
            "[--repeat=<times to repeat tests>]"
            "[--quick]"
            "[--v] "
            "[--cnp]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());
    printf("\n");

    if (quick)
    {
        // Quick test
        if (num_items < 0) num_items = 32000000;

        Test<char>(        num_items * 4, RANDOM, Sum(), int(0), CUB_TYPE_STRING(int));
        Test<short>(       num_items * 2, RANDOM, Sum(), short(0), CUB_TYPE_STRING(short));
        Test<int>(         num_items    , RANDOM, Sum(), (int) (0), CUB_TYPE_STRING(int));
        Test<long long>(   num_items / 2, RANDOM, Sum(), (long long) (0), CUB_TYPE_STRING(long long));
        Test<TestFoo>(     num_items / 4, RANDOM, Sum(), TestFoo(), CUB_TYPE_STRING(TestFoo));
    }
    else
    {
        // Repeat test sequence
        for (int i = 0; i <= g_repeat; ++i)
        {
            // Test different input types
            Test<unsigned char>(num_items, CUB_TYPE_STRING(unsigned char));
            Test<unsigned short>(num_items, CUB_TYPE_STRING(unsigned short));
            Test<unsigned int>(num_items, CUB_TYPE_STRING(unsigned int));
            Test<unsigned long long>(num_items, CUB_TYPE_STRING(unsigned long long));

            Test<uchar2>(num_items, CUB_TYPE_STRING(uchar2));
            Test<uint2>(num_items, CUB_TYPE_STRING(uint2));
            Test<ulonglong2>(num_items, CUB_TYPE_STRING(ulonglong2));
            Test<ulonglong4>(num_items, CUB_TYPE_STRING(ulonglong4));

            Test<TestFoo>(num_items, CUB_TYPE_STRING(TestFoo));
            Test<TestBar>(num_items, CUB_TYPE_STRING(TestBar));
        }
    }

    return 0;
}



