#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of DeviceScan utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <hipcub/hipcub.hpp>
#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;
int                     g_iterations = 100;
CachingDeviceAllocator  g_allocator;


//---------------------------------------------------------------------
// Dispatch to different DeviceScan entrypoints
//---------------------------------------------------------------------


/**
 * Exclusive scan specialization
 */
template <
    typename InputIteratorRA,
    typename OutputIteratorRA,
    typename ScanOp,
    typename Identity,
    typename SizeT>
__host__ __device__ __forceinline__
hipError_t Dispatch(
    void                        *d_temp_storage,                    ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
    size_t                      &temp_storage_bytes,                ///< [in,out] Size in bytes of \t d_temp_storage allocation.
    InputIteratorRA             d_in,
    OutputIteratorRA            d_out,
    ScanOp                      scan_op,
    Identity                    identity,
    SizeT                       num_items,
    hipStream_t                stream              = 0,
    bool                        stream_synchronous  = false)
{
    return DeviceScan::ExclusiveScan(d_temp_storage, temp_storage_bytes, d_in, d_out, scan_op, identity, num_items, stream, stream_synchronous);
}


/**
 * Exclusive sum specialization
 */
template <
    typename InputIteratorRA,
    typename OutputIteratorRA,
    typename T,
    typename Identity,
    typename SizeT>
__host__ __device__ __forceinline__
hipError_t Dispatch(
    void                        *d_temp_storage,                    ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
    size_t                      &temp_storage_bytes,                ///< [in,out] Size in bytes of \t d_temp_storage allocation.
    InputIteratorRA             d_in,
    OutputIteratorRA            d_out,
    Sum                         scan_op,
    Identity                    identity,
    SizeT                       num_items,
    hipStream_t                stream              = 0,
    bool                        stream_synchronous  = false)
{
    return DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, stream, stream_synchronous);
}


/**
 * Inclusive scan specialization
 */
template <
    typename InputIteratorRA,
    typename OutputIteratorRA,
    typename ScanOp,
    typename SizeT>
__host__ __device__ __forceinline__
hipError_t Dispatch(
    void                        *d_temp_storage,                    ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
    size_t                      &temp_storage_bytes,                ///< [in,out] Size in bytes of \t d_temp_storage allocation.
    InputIteratorRA             d_in,
    OutputIteratorRA            d_out,
    ScanOp                      scan_op,
    NullType                    identity,
    SizeT                       num_items,
    hipStream_t                stream              = 0,
    bool                        stream_synchronous  = false)
{
    return DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_in, d_out, scan_op, num_items, stream, stream_synchronous);
}


/**
 * Inclusive sum specialization
 */
template <
    typename InputIteratorRA,
    typename OutputIteratorRA,
    typename T,
    typename SizeT>
__host__ __device__ __forceinline__
hipError_t Dispatch(
    void                        *d_temp_storage,                    ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
    size_t                      &temp_storage_bytes,                ///< [in,out] Size in bytes of \t d_temp_storage allocation.
    InputIteratorRA             d_in,
    OutputIteratorRA            d_out,
    Sum                         scan_op,
    NullType                    identity,
    SizeT                       num_items,
    hipStream_t                stream              = 0,
    bool                        stream_synchronous  = false)
{
    return DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, stream, stream_synchronous);
}

//---------------------------------------------------------------------
// CUDA Nested Parallelism Test Kernel
//---------------------------------------------------------------------

/**
 * Simple wrapper kernel to invoke DeviceScan
 */
/*
template <
    bool                STREAM_SYNCHRONOUS,
    typename            InputIteratorRA,
    typename            OutputIteratorRA,
    typename            ScanOp>
__global__ void CnpScan(
    InputIteratorRA     d_in,
    OutputIteratorRA    d_out,
    int                 num_items,
    ScanOp              scan_op,
    int                 iterations,
    hipError_t*        d_cnp_error)
{
    hipError_t error = hipSuccess;

#ifdef CUB_RUNTIME_ENABLED
    for (int i = 0; i < iterations; ++i)
    {
        error = DeviceScan::Scan(d_in, d_out, num_items, scan_op, 0, STREAM_SYNCHRONOUS);
    }
#else
    error = hipErrorNotSupported;
#endif

    *d_cnp_error = error;
}
*/

//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize exclusive-scan problem (and solution)
 */
template <
    typename     T,
    typename     ScanOp,
    typename     IdentityT>
T Initialize(
    int          gen_mode,
    T            *h_in,
    T            *h_reference,
    int          num_items,
    ScanOp       scan_op,
    IdentityT    identity)
{
    T inclusive = identity;
    T aggregate = identity;

    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
        h_reference[i] = inclusive;
        inclusive = scan_op(inclusive, h_in[i]);
        aggregate = scan_op(aggregate, h_in[i]);
    }

    return aggregate;
}


/**
 * Initialize inclusive-scan problem (and solution)
 */
template <
    typename     T,
    typename     ScanOp>
T Initialize(
    int          gen_mode,
    T            *h_in,
    T            *h_reference,
    int          num_items,
    ScanOp       scan_op,
    NullType)
{
    T inclusive;
    T aggregate;
    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
        if (i == 0)
        {
            inclusive = h_in[0];
            aggregate = h_in[0];
        }
        else
        {
            inclusive = scan_op(inclusive, h_in[i]);
            aggregate = scan_op(aggregate, h_in[i]);
        }
        h_reference[i] = inclusive;
    }

    return aggregate;
}




//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

/**
 * Test DeviceScan
 */
template <
    typename        T,
    typename        ScanOp,
    typename        IdentityT>
void Test(
    int             num_items,
    int             gen_mode,
    ScanOp          scan_op,
    IdentityT       identity,
    char*           type_string)
{
    int compare = 0;
    int cnp_compare = 0;

    printf("%s hipcub::DeviceScan %d items, %s %d-byte elements, gen-mode %d\n\n",
        (Equals<IdentityT, NullType>::VALUE) ? "Inclusive" : "Exclusive",
        num_items,
        type_string,
        (int) sizeof(T),
        gen_mode);
    fflush(stdout);

    // Allocate host arrays
    T*  h_in = new T[num_items];
    T*  h_reference = new T[num_items];

    // Initialize problem
    Initialize(gen_mode, h_in, h_reference, num_items, scan_op, identity);

    // Allocate device arrays
    T*              d_in = NULL;
    T*              d_out = NULL;
    hipError_t*    d_cnp_error = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in,          sizeof(T) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out,         sizeof(T) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_cnp_error,   sizeof(hipError_t) * 1));

    // Initialize device arrays
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_items, hipMemcpyHostToDevice));
    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * num_items));

    // Allocate temporary storage
    void            *d_temporary_storage = NULL;
    size_t          temporary_storage_bytes = 0;

    CubDebugExit(Dispatch(d_temporary_storage, temporary_storage_bytes, d_in, d_out, scan_op, identity, num_items, 0, true));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temporary_storage, temporary_storage_bytes));

    // Run warmup/correctness iteration
    printf("Host dispatch:\n"); fflush(stdout);
    CubDebugExit(Dispatch(d_temporary_storage, temporary_storage_bytes, d_in, d_out, scan_op, identity, num_items, 0, true));

    // Check for correctness (and display results, if specified)
    compare = CompareDeviceResults(h_reference, d_out, num_items, true, g_verbose);
    printf("\n%s", compare ? "FAIL" : "PASS");

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Performance
    GpuTimer gpu_timer;
    float elapsed_millis = 0.0;
    for (int i = 0; i < g_iterations; i++)
    {
        gpu_timer.Start();

        CubDebugExit(Dispatch(d_temporary_storage, temporary_storage_bytes, d_in, d_out, scan_op, identity, num_items));

        gpu_timer.Stop();
        elapsed_millis += gpu_timer.ElapsedMillis();
    }
    if (g_iterations > 0)
    {
        float avg_millis = elapsed_millis / g_iterations;
        float grate = float(num_items) / avg_millis / 1000.0 / 1000.0;
        float gbandwidth = grate * sizeof(T) * 2;
        printf(", %.3f avg ms, %.3f billion items/s, %.3f GB/s\n", avg_millis, grate, gbandwidth);
    }
    else
    {
        printf("\n");
    }


    // Evaluate using CUDA nested parallelism
/*
#if (TEST_CNP == 1)

    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * 1));

    // Run warmup/correctness iteration
    printf("\nDevice dispatch:\n"); fflush(stdout);
    CnpScan<true><<<1,1>>>(d_in, d_out, num_items, scan_op, 1, d_cnp_error);

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Check if we were compiled and linked for CNP
    hipError_t h_cnp_error;
    CubDebugExit(hipMemcpy(&h_cnp_error, d_cnp_error, sizeof(hipError_t) * 1, hipMemcpyDeviceToHost));
    if (h_cnp_error == hipErrorInvalidConfiguration)
    {
        printf("CNP not supported");
    }
    else
    {
        CubDebugExit(h_cnp_error);

        // Check for correctness (and display results, if specified)
        cnp_compare = CompareDeviceResults(h_reference, d_out, 1, g_verbose, g_verbose);
        printf("\n%s", cnp_compare ? "FAIL" : "PASS");

        // Performance
        gpu_timer.Start();

        CnpScan<false><<<1,1>>>(d_in, d_out, num_items, scan_op, g_iterations, d_cnp_error);

        gpu_timer.Stop();
        elapsed_millis = gpu_timer.ElapsedMillis();

        if (g_iterations > 0)
        {
            float avg_millis = elapsed_millis / g_iterations;
            float grate = float(num_items) / avg_millis / 1000.0 / 1000.0;
            float gbandwidth = grate * sizeof(T) * 2;
            printf(", %.3f avg ms, %.3f billion items/s, %.3f GB/s\n", avg_millis, grate, gbandwidth);
        }
        else
        {
            printf("\n");
        }
    }

#endif
*/

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_cnp_error) CubDebugExit(g_allocator.DeviceFree(d_cnp_error));
    if (d_temporary_storage) CubDebugExit(g_allocator.DeviceFree(d_temporary_storage));

    // Correctness asserts
    AssertEquals(0, compare);
    AssertEquals(0, cnp_compare);
}




//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------




/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_items = 1 * 1024 * 1024;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    args.GetCmdLineArgument("n", num_items);
    args.GetCmdLineArgument("i", g_iterations);
    g_verbose = args.CheckCmdLineFlag("v");
    bool quick = args.CheckCmdLineFlag("quick");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--v] "
            "[--cnp]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Quick test
    typedef int T;
    Test<T>(num_items, UNIFORM, Sum(), T(0), CUB_TYPE_STRING(T));



/*
    // primitives
    Test<char>(Sum<char>(), CUB_TYPE_STRING(char));
    Test<short>(Sum<short>(), CUB_TYPE_STRING(short));
    Test<int>(Sum<int>(), CUB_TYPE_STRING(int));
    Test<long long>(Sum<long long>(), CUB_TYPE_STRING(long long));

    // vector types
    Test<char2>(Sum<char2>(), CUB_TYPE_STRING(char2));
    Test<short2>(Sum<short2>(), CUB_TYPE_STRING(short2));
    Test<int2>(Sum<int2>(), CUB_TYPE_STRING(int2));
    Test<longlong2>(Sum<longlong2>(), CUB_TYPE_STRING(longlong2));

    Test<char4>(Sum<char4>(), CUB_TYPE_STRING(char4));
    Test<short4>(Sum<short4>(), CUB_TYPE_STRING(short4));
    Test<int4>(Sum<int4>(), CUB_TYPE_STRING(int4));
    Test<longlong4>(Sum<longlong4>(), CUB_TYPE_STRING(longlong4));

    // Complex types
    Test<TestFoo>(Sum<TestFoo>(), CUB_TYPE_STRING(TestFoo));
    Test<TestBar>(Sum<TestBar>(), CUB_TYPE_STRING(TestBar));
*/
    return 0;
}



