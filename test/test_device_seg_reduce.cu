#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * An implementation of segmented reduction using a load-balanced parallelization
 * strategy based on the MergePath decision path.
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iterator>
#include <vector>
#include <string>
#include <algorithm>
#include <stdio.h>

#include <hipcub/hipcub.hpp>

#include "test_util.h"

using namespace hipcub;
using namespace std;


/******************************************************************************
 * Globals, constants, and typedefs
 ******************************************************************************/

bool                    g_verbose           = false;
int                     g_timing_iterations = 1;
CachingDeviceAllocator  g_allocator(true);


/******************************************************************************
 * Utility routines
 ******************************************************************************/


/**
 * Computes the begin offsets into A and B for the specified
 * location (diagonal) along the merge decision path
 */
template <
    typename    IteratorA,
    typename    IteratorB,
    typename    Offset>
__device__ __forceinline__ void MergePathSearch(
    Offset      diagonal,
    IteratorA   a,
    Offset      a_begin,
    Offset      a_end,
    Offset      &a_offset,
    IteratorB   b,
    Offset      b_begin,
    Offset      b_end,
    Offset      &b_offset)
{
    Offset split_min = CUB_MAX(diagonal - b_end, a_begin);
    Offset split_max = CUB_MIN(diagonal, a_end);

    while (split_min < split_max)
    {
        Offset split_pivot = (split_min + split_max) >> 1;
        if (a[split_pivot] <= b[diagonal - split_pivot - 1])
        {
            // Move candidate split range up A, down B
            split_min = split_pivot + 1;
        }
        else
        {
            // Move candidate split range up B, down A
            split_max = split_pivot;
        }
    }

    a_offset = CUB_MIN(split_min, a_end);
    b_offset = CUB_MIN(diagonal - split_min, b_end);
}

/**
 * Computes the begin offsets into A and B for the specified
 * location (diagonal) along the merge decision path
 */
template <
    typename    IteratorA,
    typename    IteratorB,
    typename    Offset>
__device__ __forceinline__ void MergePathSearch2(
    Offset      diagonal,
    IteratorA   a,
    Offset      a_begin,
    Offset      a_end,
    Offset      &a_offset,
    IteratorB   b,
    Offset      b_begin,
    Offset      b_end,
    Offset      &b_offset)
{
    Offset split_min = CUB_MAX(diagonal - b_end, a_begin);
    Offset split_max = CUB_MIN(diagonal, a_end);

    while (split_min < split_max)
    {
        Offset split_pivot = (split_min + split_max) >> 1;
        if (a[split_pivot] <= b[diagonal - split_pivot - 1])
        {
            // Move candidate split range up A, down B
            split_min = split_pivot + 1;
        }
        else
        {
            // Move candidate split range up B, down A
            split_max = split_pivot;
        }
    }

    a_offset = CUB_MIN(split_min, a_end);
    b_offset = CUB_MIN(diagonal - split_min, b_end);
}


/******************************************************************************
 * Tuning policy types
 ******************************************************************************/

/**
 * Parameterizable tuning policy type for BlockSegReduceRegion
 */
template <
    int                     _BLOCK_THREADS,             ///< Threads per thread block
    int                     _ITEMS_PER_THREAD,          ///< Items per thread (per tile of input)
    bool                    _USE_SMEM_SEGMENT_CACHE,    ///< Whether or not to cache incoming segment offsets in shared memory before reducing each tile
    bool                    _USE_SMEM_VALUE_CACHE,      ///< Whether or not to cache incoming values in shared memory before reducing each tile
    CacheLoadModifier       _LOAD_MODIFIER_SEGMENTS,    ///< Cache load modifier for reading segment offsets
    CacheLoadModifier       _LOAD_MODIFIER_VALUES,      ///< Cache load modifier for reading values
    BlockReduceAlgorithm    _REDUCE_ALGORITHM,          ///< The BlockReduce algorithm to use
    BlockScanAlgorithm      _SCAN_ALGORITHM>            ///< The BlockScan algorithm to use
struct BlockSegReduceRegionPolicy
{
    enum
    {
        BLOCK_THREADS           = _BLOCK_THREADS,               ///< Threads per thread block
        ITEMS_PER_THREAD        = _ITEMS_PER_THREAD,            ///< Items per thread (per tile of input)
        USE_SMEM_SEGMENT_CACHE  = _USE_SMEM_SEGMENT_CACHE,      ///< Whether or not to cache incoming segment offsets in shared memory before reducing each tile
        USE_SMEM_VALUE_CACHE    = _USE_SMEM_VALUE_CACHE,        ///< Whether or not to cache incoming upcoming values in shared memory before reducing each tile
    };

    static const CacheLoadModifier      LOAD_MODIFIER_SEGMENTS  = _LOAD_MODIFIER_SEGMENTS;  ///< Cache load modifier for reading segment offsets
    static const CacheLoadModifier      LOAD_MODIFIER_VALUES    = _LOAD_MODIFIER_VALUES;    ///< Cache load modifier for reading values
    static const BlockReduceAlgorithm   REDUCE_ALGORITHM        = _REDUCE_ALGORITHM;        ///< The BlockReduce algorithm to use
    static const BlockScanAlgorithm     SCAN_ALGORITHM          = _SCAN_ALGORITHM;          ///< The BlockScan algorithm to use
};


/******************************************************************************
 * Persistent thread block types
 ******************************************************************************/

/**
 * \brief BlockSegReduceTiles implements a stateful abstraction of CUDA thread blocks for participating in device-wide segmented reduction.
 */
template <
    typename BlockSegReduceRegionPolicy,    ///< Parameterized BlockSegReduceRegionPolicy tuning policy
    typename SegmentOffsetIterator,         ///< Random-access input iterator type for reading segment end-offsets
    typename ValueIterator,                 ///< Random-access input iterator type for reading values
    typename OutputIterator,                ///< Random-access output iterator type for writing segment reductions
    typename ReductionOp,                   ///< Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
    typename Offset>                        ///< Signed integer type for global offsets
struct BlockSegReduceRegion
{
    //---------------------------------------------------------------------
    // Types and constants
    //---------------------------------------------------------------------

    // Constants
    enum
    {
        BLOCK_THREADS       = BlockSegReduceRegionPolicy::BLOCK_THREADS,
        ITEMS_PER_THREAD    = BlockSegReduceRegionPolicy::ITEMS_PER_THREAD,
        TILE_ITEMS          = BLOCK_THREADS * ITEMS_PER_THREAD,                     /// Number of work items to be processed per tile
    };

    // Segment offset type
    typedef typename std::iterator_traits<SegmentOffsetIterator>::value_type SegmentOffset;

    // Value type
    typedef typename std::iterator_traits<ValueIterator>::value_type Value;

    // Counting iterator type
    typedef CountingInputIterator<SegmentOffset, Offset> CountingIterator;

    // Segment offsets iterator wrapper type
    typedef typename If<(IsPointer<SegmentOffsetIterator>::VALUE),
            CacheModifiedInputIterator<BlockSegReduceRegionPolicy::LOAD_MODIFIER_SEGMENTS, SegmentOffset, Offset>,  // Wrap the native input pointer with CacheModifiedInputIterator
            SegmentOffsetIterator>::Type                                                                            // Directly use the supplied input iterator type
        WrappedSegmentOffsetIterator;

    // Values iterator wrapper type
    typedef typename If<(IsPointer<ValueIterator>::VALUE),
            CacheModifiedInputIterator<BlockSegReduceRegionPolicy::LOAD_MODIFIER_VALUES, Value, Offset>,        // Wrap the native input pointer with CacheModifiedInputIterator
            ValueIterator>::Type                                                                                // Directly use the supplied input iterator type
        WrappedValueIterator;

    // Tail flag type for marking segment discontinuities
    typedef int TailFlag;

    // Reduce-by-key data type tuple (segment-ID, value)
    typedef KeyValuePair<Offset, Value> KeyValuePair;

    // BlockScan scan operator for reduction-by-segment
    typedef ReduceByKeyOp<ReductionOp> ReduceByKeyOp;

    // Stateful BlockScan prefix callback type for managing a running total while scanning consecutive tiles
    typedef RunningBlockPrefixCallbackOp<
            KeyValuePair,
            ReduceByKeyOp>
        RunningPrefixCallbackOp;

    // Parameterized BlockReduce type for block-wide reduction
    typedef BlockReduce<
            Value,
            BLOCK_THREADS,
            BlockSegReduceRegionPolicy::REDUCE_ALGORITHM>
        BlockReduce;

    // Parameterized BlockScan type for block-wide reduce-value-by-key
    typedef BlockScan<
            KeyValuePair,
            BLOCK_THREADS,
            BlockSegReduceRegionPolicy::SCAN_ALGORITHM>
        BlockScan;

    // Shared memory type for this threadblock
    struct _TempStorage
    {
        union
        {
            // Smem needed for BlockScan
            typename BlockScan::TempStorage     scan;

            // Smem needed for BlockReduce
            typename BlockReduce::TempStorage   reduce;

            struct
            {
                // Smem needed for communicating start/end indices between threads for a given work tile
                Offset thread_segment_idx[BLOCK_THREADS + 1];
                Value thread_value_idx[BLOCK_THREADS + 1];
            };
        };

        Offset block_segment_idx[2];     // The starting and ending indices of segment offsets for the threadblock's region
        Offset block_value_idx[2];       // The starting and ending indices of values for the threadblock's region

        // The first partial reduction tuple scattered by this thread block
        KeyValuePair first_tuple;
    };


    // Alias wrapper allowing storage to be unioned
    struct TempStorage : Uninitialized<_TempStorage> {};


    //---------------------------------------------------------------------
    // Thread fields
    //---------------------------------------------------------------------

    _TempStorage                    &temp_storage;          ///< Reference to shared storage
    WrappedSegmentOffsetIterator    d_segment_end_offsets;  ///< A sequence of \p num_segments segment end-offsets
    WrappedValueIterator            d_values;               ///< A sequence of \p num_values data to reduce
    OutputIterator                  d_output;               ///< A sequence of \p num_segments segment totals
    CountingIterator                d_value_offsets;        ///< A sequence of \p num_values value-offsets
    Offset                          num_values;             ///< Total number of values to reduce
    Offset                          num_segments;           ///< Number of segments being reduced
    Value                           identity;               ///< Identity value (for zero-length segments)
    ReductionOp                     reduction_op;           ///< Reduction operator
    ReduceByKeyOp                   scan_op;                ///< Reduce-by-key scan operator
    RunningPrefixCallbackOp         prefix_op;              ///< Stateful running total for block-wide prefix scan of partial reduction tuples


    //---------------------------------------------------------------------
    // Operations
    //---------------------------------------------------------------------

    /**
     * Constructor
     */
    __device__ __forceinline__
    BlockSegReduceRegion(
        TempStorage             &temp_storage,          ///< Reference to shared storage
        SegmentOffsetIterator   d_segment_end_offsets,  ///< A sequence of \p num_segments segment end-offsets
        ValueIterator           d_values,               ///< A sequence of \p num_values values
        OutputIterator          d_output,               ///< A sequence of \p num_segments segment totals
        Offset                  num_values,             ///< Number of values to reduce
        Offset                  num_segments,           ///< Number of segments being reduced
        Value                   identity,               ///< Identity value (for zero-length segments)
        ReductionOp             reduction_op)           ///< Reduction operator
    :
        temp_storage(temp_storage.Alias()),
        d_segment_end_offsets(d_segment_end_offsets),
        d_values(d_values),
        d_value_offsets(0),
        d_output(d_output),
        num_values(num_values),
        num_segments(num_segments),
        identity(identity),
        reduction_op(reduction_op),
        scan_op(reduction_op),
        prefix_op(scan_op)
    {}


    /**
     * Have the thread block process the specified region of the MergePath decision path
     */
    __device__ __forceinline__ void ProcessBlockRegion(
        Offset          block_diagonal,
        Offset          next_block_diagonal,
        KeyValuePair    &first_tuple,       // [Out] Valid in thread-0
        KeyValuePair    &last_tuple)        // [Out] Valid in thread-0
    {
        // Thread block initialization
        if (threadIdx.x < 2)
        {
            Offset diagonal = (threadIdx.x == 0) ?
                block_diagonal :        // First thread searches for start indices
                next_block_diagonal;    // Second thread searches for end indices

            // Search for block starting and ending indices
            Offset block_segment_idx;
            Offset block_value_idx;

            MergePathSearch(
                diagonal,               // Diagonal
                d_segment_end_offsets,  // A (segment end-offsets)
                0,                      // Start index into A
                num_segments,           // End index into A
                block_segment_idx,      // [out] Block index into A
                d_value_offsets,        // B (value offsets)
                0,                      // Start index into B
                num_values,             // End index into B
                block_value_idx);       // [out] Block index into B

            // Share block starting and ending indices
            temp_storage.block_segment_idx[threadIdx.x] = block_segment_idx;
            temp_storage.block_value_idx[threadIdx.x] = block_value_idx;

            // Initialize the block's running prefix
            if (threadIdx.x == 0)
            {
                prefix_op.running_total.key    = block_segment_idx;
                prefix_op.running_total.value  = identity;

                // Initialize the "first scattered partial reduction tuple" to the prefix tuple (in case we don't actually scatter one)
                temp_storage.first_tuple = prefix_op.running_total;
            }
        }

        // Ensure coherence of region indices
        __syncthreads();

        // Read block's starting indices
        Offset block_segment_idx        = temp_storage.block_segment_idx[0];
        Offset block_value_idx          = temp_storage.block_value_idx[0];

        // Have the thread block iterate over the region
        #pragma unroll 1
        while (block_diagonal < next_block_diagonal)
        {
            // Read block's ending indices (hoist?)
            Offset next_block_segment_idx   = temp_storage.block_segment_idx[1];
            Offset next_block_value_idx     = temp_storage.block_value_idx[1];
/*
            if (threadIdx.x == 0) CubLog("block diagonal %d next diagonal %d, segment idx [%d: %d], value idx [%d : %d]\n",
                block_diagonal,
                next_block_diagonal,
                block_segment_idx,
                next_block_segment_idx,
                block_value_idx,
                next_block_value_idx);
*/
            // Clamp the per-thread search range to within one work-tile of block's current indices
            Offset next_tile_segment_idx    = CUB_MIN(next_block_segment_idx,   block_segment_idx + TILE_ITEMS);
            Offset next_tile_value_idx      = CUB_MIN(next_block_value_idx,     block_value_idx + TILE_ITEMS);

            // Have each thread search for the end-indices of its subranges within the segment and value inputs
            Offset next_thread_diagonal = block_diagonal + ((threadIdx.x + 1) * ITEMS_PER_THREAD);
            Offset next_thread_segment_idx;
            Offset next_thread_value_idx;

            MergePathSearch2(
                next_thread_diagonal,           // Next thread diagonal
                d_segment_end_offsets,          // A (segment end-offsets)
                block_segment_idx,              // Start index into A
                next_tile_segment_idx,          // End index into A
                next_thread_segment_idx,        // [out] Thread index into A
                d_value_offsets,                // B (value offsets)
                block_value_idx,                // Start index into B
                next_tile_value_idx,            // End index into B
                next_thread_value_idx);         // [out] Thread index into B

            // Share thread end-indices
            temp_storage.thread_segment_idx[threadIdx.x + 1]   = next_thread_segment_idx;
            temp_storage.thread_value_idx[threadIdx.x + 1]     = next_thread_value_idx;

            // Ensure coherence of search indices
            __syncthreads();

            // Retrieve the block's starting indices for the next tile of work (i.e., the last thread's end-indices)
            next_tile_segment_idx   = temp_storage.thread_segment_idx[BLOCK_THREADS];
            next_tile_value_idx     = temp_storage.thread_value_idx[BLOCK_THREADS];

            if (block_segment_idx == next_tile_segment_idx)
            {
                // There are no segment end-offsets in this tile.  Perform a
                // simple block-wide reduction and accumulate the result into
                // the running total.

                // Load a tile's worth of values (using identity for out-of-bounds items)
                Value values[ITEMS_PER_THREAD];
                Offset num_values = next_tile_value_idx - block_value_idx;
                LoadStriped<BLOCK_THREADS>(threadIdx.x, d_values + block_value_idx, values, num_values, identity);

                // Barrier for smem reuse
                __syncthreads();

                // Reduce the tile of values and update the running total in thread-0
                KeyValuePair tile_aggregate;
                tile_aggregate.key = block_segment_idx;
                tile_aggregate.value = BlockReduce(temp_storage.reduce).Reduce(values, reduction_op);

                if (threadIdx.x == 0)
                {
                    prefix_op.running_total = scan_op(
                        prefix_op.running_total,
                        tile_aggregate);
                }
            }
/*            else if (block_value_idx == next_tile_value_idx)
            {
                // There are no values in this tile (only empty segments).  Write
                // out a tile of identity values to output.

                Value segment_reductions[ITEMS_PER_THREAD];

                if (threadIdx.x == 0)
                {
                    // The first segment gets the running segment total
                    segment_reductions[0] = prefix_op.running_total.value;

                    // Update the running prefix
                    prefix_op.running_total.value = identity;
                    prefix_op.running_total.key = next_tile_segment_idx;
                }
                else
                {
                    // Remainder of segments in this tile get identity
                    segment_reductions[0] = identity;
                }

                // Remainder of segments in this tile get identity
                #pragma unroll
                for (int ITEM = 1; ITEM < ITEMS_PER_THREAD; ++ITEM)
                    segment_reductions[ITEM] = identity;

                // Store reductions
                Offset num_segments = next_tile_segment_idx - block_segment_idx;
                StoreStriped<BLOCK_THREADS>(threadIdx.x, d_output + block_segment_idx, segment_reductions, num_segments);
            }
*/            else
            {
                // Merge the tile's segment and value indices

                // Get thread begin-index for segments
                Offset thread_segment_idx = (threadIdx.x == 0) ?
                    block_segment_idx :                             // First thread starts at the block's start
                    temp_storage.thread_segment_idx[threadIdx.x];   // Other threads start at their predecessor's end

                // Get thread begin-index for values
                Offset thread_value_idx = (threadIdx.x == 0) ?
                    block_value_idx :                               // First thread starts at the block's start
                    temp_storage.thread_value_idx[threadIdx.x];     // Other threads start at their predecessor's end
/*
                CubLog("\t thread segment idx %d:%d, value idx %d:%d\n",
                    thread_segment_idx, next_thread_segment_idx,
                    thread_value_idx, next_thread_value_idx);
*/
                // Barrier for smem reuse
                __syncthreads();

                // Check if first segment end-offset is in range
                bool valid_segment = (thread_segment_idx < next_thread_segment_idx);

                // Check if first value offset is in range
                bool valid_value = (thread_value_idx < next_thread_value_idx);

                // Load first segment end-offset
                Offset segment_end_offset = (valid_segment) ?
                    d_segment_end_offsets[thread_segment_idx] :
                    num_values;                                                     // Out of range (the last segment end-offset is one-past the last value offset)

                // Load first value offset
                Offset value_offset = (valid_value) ?
                    d_value_offsets[thread_value_idx] :
                    num_values;                                                     // Out of range (one-past the last value offset)

                // Assemble segment-demarcating tail flags and partial reduction tuples
                TailFlag        tail_flags[ITEMS_PER_THREAD];
                KeyValuePair    partial_reductions[ITEMS_PER_THREAD];

                #pragma unroll
                for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
                {
                    // Default tuple and flag values
                    partial_reductions[ITEM].key    = thread_segment_idx;
                    partial_reductions[ITEM].value  = identity;
                    tail_flags[ITEM]                = 0;

                    // Whether or not we slide (a) right along the segment path or (b) down the value path
                    if (valid_segment && (!valid_value || (segment_end_offset <= value_offset)))
                    {
                        // Consume this segment index

                        // Set tail flag noting the end of the segment
                        tail_flags[ITEM] = 1;

                        // Increment segment index
                        thread_segment_idx++;

                        // Read next segment end-offset (if valid)
                        if ((valid_segment = (thread_segment_idx < next_thread_segment_idx)))
                            segment_end_offset = d_segment_end_offsets[thread_segment_idx];
                    }
                    else if (valid_value)
                    {
                        // Consume this value index

                        // Update the tuple's value with the value at this index.
                        partial_reductions[ITEM].value = d_values[thread_value_idx];

                        // Increment value index
                        thread_value_idx++;

                        // Read next value offset (if valid)
                        if ((valid_value = (thread_value_idx < next_thread_value_idx)))
                            value_offset = d_value_offsets[thread_value_idx];
                    }
                }

/*
                CubLog("Tuples %s<%d,%.1f>, %s<%d,%.1f>\n",
                    tail_flags[0] ? "*" : "", partial_reductions[0].key, partial_reductions[0].value,
                    tail_flags[1] ? "*" : "", partial_reductions[1].key, partial_reductions[1].value);
*/
                // Use prefix scan to reduce values by segment-id.  The segment-reductions end up in items flagged as segment-tails.
                KeyValuePair block_aggregate;
                BlockScan(temp_storage.scan).InclusiveScan(
                    partial_reductions,             // Scan input
                    partial_reductions,             // Scan output
                    scan_op,                        // Scan operator
                    block_aggregate,                // Block-wide total (unused)
                    prefix_op);                     // Prefix operator for seeding the block-wide scan with the running total

/*
                CubLog("\t\t Scanned tuples %s<%d,%.1f>, %s<%d,%.1f>\n",
                    tail_flags[0] ? "*" : "", partial_reductions[0].key, partial_reductions[0].value,
                    tail_flags[1] ? "*" : "", partial_reductions[1].key, partial_reductions[1].value);
*/
                // The first segment index for this region (hoist?)
                Offset first_segment_idx = temp_storage.block_segment_idx[0];

                // Scatter an accumulated reduction if it is the head of a valid segment
                #pragma unroll
                for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++)
                {
                    if (tail_flags[ITEM])
                    {
                        Offset  segment_idx = partial_reductions[ITEM].key;
                        Value   value       = partial_reductions[ITEM].value;

                        // Write value reduction to corresponding segment id
                        d_output[segment_idx] = value;

                        // Save off the first value product that this thread block will scatter
                        if (segment_idx == first_segment_idx)
                        {
                            temp_storage.first_tuple.value = value;
                        }
                    }
                }
            }


            // Advance to the next region in the decision path
            block_diagonal += TILE_ITEMS;

            // Advance the block's indices in preparation for the next tile
            block_segment_idx   = next_tile_segment_idx;
            block_value_idx     = next_tile_value_idx;

            // Barrier for smem reuse
            __syncthreads();
        }

        // Get first and last tuples for the region
        if (threadIdx.x == 0)
        {
            first_tuple = temp_storage.first_tuple;
            last_tuple = prefix_op.running_total;
        }
    }


};








/******************************************************************************
 * Tuning policy types
 ******************************************************************************/

/**
 * Parameterizable tuning policy type for BlockSegReduceRegionByKey
 */
template <
    int                     _BLOCK_THREADS,             ///< Threads per thread block
    int                     _ITEMS_PER_THREAD,          ///< Items per thread (per tile of input)
    BlockLoadAlgorithm      _LOAD_ALGORITHM,            ///< The BlockLoad algorithm to use
    bool                    _LOAD_WARP_TIME_SLICING,    ///< Whether or not only one warp's worth of shared memory should be allocated and time-sliced among block-warps during any load-related data transpositions (versus each warp having its own storage)
    CacheLoadModifier       _LOAD_MODIFIER,             ///< Cache load modifier for reading input elements
    BlockScanAlgorithm      _SCAN_ALGORITHM>            ///< The BlockScan algorithm to use
struct BlockSegReduceRegionByKeyPolicy
{
    enum
    {
        BLOCK_THREADS           = _BLOCK_THREADS,               ///< Threads per thread block
        ITEMS_PER_THREAD        = _ITEMS_PER_THREAD,            ///< Items per thread (per tile of input)
        LOAD_WARP_TIME_SLICING  = _LOAD_WARP_TIME_SLICING,      ///< Whether or not only one warp's worth of shared memory should be allocated and time-sliced among block-warps during any load-related data transpositions (versus each warp having its own storage)    };
    };

    static const BlockLoadAlgorithm     LOAD_ALGORITHM          = _LOAD_ALGORITHM;      ///< The BlockLoad algorithm to use
    static const CacheLoadModifier      LOAD_MODIFIER           = _LOAD_MODIFIER;       ///< Cache load modifier for reading input elements
    static const BlockScanAlgorithm     SCAN_ALGORITHM          = _SCAN_ALGORITHM;      ///< The BlockScan algorithm to use
};


/******************************************************************************
 * Persistent thread block types
 ******************************************************************************/

/**
 * \brief BlockSegReduceRegionByKey implements a stateful abstraction of CUDA thread blocks for participating in device-wide reduce-value-by-key.
 */
template <
    typename    BlockSegReduceRegionByKeyPolicy,        ///< Parameterized BlockSegReduceRegionByKeyPolicy tuning policy
    typename    InputIterator,                          ///< Random-access iterator referencing key-value input tuples
    typename    OutputIterator,                         ///< Random-access iterator referencing segment output totals
    typename    ReductionOp>                            ///< Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
struct BlockSegReduceRegionByKey
{
    //---------------------------------------------------------------------
    // Types and constants
    //---------------------------------------------------------------------

    // Constants
    enum
    {
        BLOCK_THREADS       = BlockSegReduceRegionByKeyPolicy::BLOCK_THREADS,
        ITEMS_PER_THREAD    = BlockSegReduceRegionByKeyPolicy::ITEMS_PER_THREAD,
        TILE_ITEMS          = BLOCK_THREADS * ITEMS_PER_THREAD,
    };

    // KeyValuePair input type
    typedef typename std::iterator_traits<InputIterator>::value_type KeyValuePair;

    // Signed integer type for global offsets
    typedef typename KeyValuePair::Key Offset;

    // Value type
    typedef typename KeyValuePair::Value Value;

    // Head flag type
    typedef int HeadFlag;

    // Input iterator wrapper type for loading KeyValuePair elements through cache
    typedef CacheModifiedInputIterator<
            BlockSegReduceRegionByKeyPolicy::LOAD_MODIFIER,
            KeyValuePair,
            Offset>
        WrappedInputIterator;

    // Parameterized BlockLoad type
    typedef BlockLoad<
            WrappedInputIterator,
            BLOCK_THREADS,
            ITEMS_PER_THREAD,
            BlockSegReduceRegionByKeyPolicy::LOAD_ALGORITHM,
            BlockSegReduceRegionByKeyPolicy::LOAD_WARP_TIME_SLICING>
        BlockLoad;

    // BlockScan scan operator for reduction-by-segment
    typedef ReduceByKeyOp<ReductionOp> ReduceByKeyOp;

    // Stateful BlockScan prefix callback type for managing a running total while scanning consecutive tiles
    typedef RunningBlockPrefixCallbackOp<
            KeyValuePair,
            ReduceByKeyOp>
        RunningPrefixCallbackOp;

    // Parameterized BlockScan type for block-wide reduce-value-by-key
    typedef BlockScan<
            KeyValuePair,
            BLOCK_THREADS,
            BlockSegReduceRegionByKeyPolicy::SCAN_ALGORITHM>
        BlockScan;

    // Parameterized BlockDiscontinuity type for identifying key discontinuities
    typedef BlockDiscontinuity<
            Offset,
            BLOCK_THREADS>
        BlockDiscontinuity;

    // Operator for detecting discontinuities in a list of segment identifiers.
    struct NewSegmentOp
    {
        /// Returns true if row_b is the start of a new row
        __device__ __forceinline__ bool operator()(const Offset& b, const Offset& a)
        {
            return (a != b);
        }
    };

    // Shared memory type for this threadblock
    struct _TempStorage
    {
        union
        {
            typename BlockLoad::TempStorage                 load;           // Smem needed for tile loading
            struct {
                typename BlockScan::TempStorage             scan;           // Smem needed for reduce-value-by-segment scan
                typename BlockDiscontinuity::TempStorage    discontinuity;  // Smem needed for head-flagging
            };
        };
    };

    // Alias wrapper allowing storage to be unioned
    struct TempStorage : Uninitialized<_TempStorage> {};


    //---------------------------------------------------------------------
    // Thread fields
    //---------------------------------------------------------------------

    _TempStorage                &temp_storage;          ///< Reference to shared storage
    WrappedInputIterator        d_tuple_partials;       ///< A sequence of partial reduction tuples to scan
    OutputIterator              d_output;               ///< A sequence of segment totals
    Value                       identity;               ///< Identity value (for zero-length segments)
    ReduceByKeyOp               scan_op;                ///< Reduce-by-key scan operator
    RunningPrefixCallbackOp     prefix_op;              ///< Stateful running total for block-wide prefix scan of partial reduction tuples


    //---------------------------------------------------------------------
    // Operations
    //---------------------------------------------------------------------

    /**
     * Constructor
     */
    __device__ __forceinline__
    BlockSegReduceRegionByKey(
        TempStorage             &temp_storage,          ///< Reference to shared storage
        InputIterator           d_tuple_partials,       ///< A sequence of partial reduction tuples to scan
        OutputIterator          d_output,               ///< A sequence of segment totals
        Value                   identity,               ///< Identity value (for zero-length segments)
        ReductionOp             reduction_op)           ///< Reduction operator
    :
        temp_storage(temp_storage.Alias()),
        d_tuple_partials(d_tuple_partials),
        d_output(d_output),
        identity(identity),
        scan_op(reduction_op),
        prefix_op(scan_op)
    {}



    /**
     * Processes a reduce-value-by-key input tile, outputting reductions for each segment
     */
    template <bool FULL_TILE>
    __device__ __forceinline__
    void ProcessTile(
        Offset block_offset,
        Offset first_segment_idx,
        Offset last_segment_idx,
        int guarded_items = TILE_ITEMS)
    {
        KeyValuePair    partial_reductions[ITEMS_PER_THREAD];
        Offset          segment_ids[ITEMS_PER_THREAD];
        HeadFlag        head_flags[ITEMS_PER_THREAD];

        // Load a tile of block partials from previous kernel
        if (FULL_TILE)
        {
            // Full tile
            BlockLoad(temp_storage.load).Load(d_tuple_partials + block_offset, partial_reductions);
        }
        else
        {
            KeyValuePair oob_default;
            oob_default.key    = last_segment_idx;       // The last segment ID to be reduced
            oob_default.value  = identity;

            // Partially-full tile
            BlockLoad(temp_storage.load).Load(d_tuple_partials + block_offset, partial_reductions, guarded_items, oob_default);
        }

        // Barrier for shared memory reuse
        __syncthreads();

        // Copy the segment IDs for head-flagging
        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++)
        {
            segment_ids[ITEM] = partial_reductions[ITEM].key;
        }

        // Flag segment heads by looking for discontinuities
        BlockDiscontinuity(temp_storage.discontinuity).FlagHeads(
            head_flags,                         // [out] Head flags
            segment_ids,                        // Segment ids
            NewSegmentOp(),                     // Functor for detecting start of new rows
            prefix_op.running_total.key);       // Last segment ID from previous tile to compare with first segment ID in this tile

        // Reduce-value-by-segment across partial_reductions using exclusive prefix scan
        KeyValuePair block_aggregate;
        BlockScan(temp_storage.scan).ExclusiveScan(
            partial_reductions,                   // Scan input
            partial_reductions,                   // Scan output
            scan_op,                        // Scan operator
            block_aggregate,                // Block-wide total (unused)
            prefix_op);                     // Prefix operator for seeding the block-wide scan with the running total
/*
        CubLog("Scanned tuples %s<%d,%.1f>, %s<%d,%.1f>\n",
            head_flags[0] ? "*" : "", partial_reductions[0].key, partial_reductions[0].value,
            head_flags[1] ? "*" : "", partial_reductions[1].key, partial_reductions[1].value);
*/
        // Scatter an accumulated reduction if it is the head of a valid segment
        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++)
        {
            if (head_flags[ITEM])
            {
                d_output[partial_reductions[ITEM].key] = partial_reductions[ITEM].value;
            }
        }
    }


    /**
     * Iterate over input tiles belonging to this thread block
     */
    __device__ __forceinline__
    void ProcessRegion(
        Offset block_offset,
        Offset block_end,
        Offset first_segment_idx,
        Offset last_segment_idx)
    {
        if (threadIdx.x == 0)
        {
            // Initialize running prefix to the first segment index paired with identity
            prefix_op.running_total.key    = first_segment_idx;
            prefix_op.running_total.value  = identity;
        }

        // Process full tiles
        while (block_offset + TILE_ITEMS <= block_end)
        {
            ProcessTile<true>(block_offset, first_segment_idx, last_segment_idx);
            __syncthreads();

            block_offset += TILE_ITEMS;
        }

        // Process final value tile (if present)
        int guarded_items = block_end - block_offset;
        if (guarded_items)
        {
            ProcessTile<false>(block_offset, first_segment_idx, last_segment_idx, guarded_items);
        }
    }
};



/******************************************************************************
 * Kernel entrypoints
 ******************************************************************************/



/**
 * Segmented reduce region kernel entry point (multi-block).
 */
template <
    typename BlockSegReduceRegionPolicy,        ///< Parameterized BlockSegReduceRegionPolicy tuning policy
    typename SegmentOffsetIterator,             ///< Random-access input iterator type for reading segment end-offsets
    typename ValueIterator,                     ///< Random-access input iterator type for reading values
    typename OutputIterator,                    ///< Random-access output iterator type for writing segment reductions
    typename ReductionOp,                       ///< Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
    typename Offset,                            ///< Signed integer type for global offsets
    typename Value>                             ///< Value type
__launch_bounds__ (BlockSegReduceRegionPolicy::BLOCK_THREADS)
__global__ void SegReduceRegionKernel(
    SegmentOffsetIterator       d_segment_end_offsets,  ///< [in] A sequence of \p num_segments segment end-offsets
    ValueIterator               d_values,               ///< [in] A sequence of \p num_values values
    OutputIterator              d_output,               ///< [out] A sequence of \p num_segments segment totals
    KeyValuePair<Offset, Value> *d_tuple_partials,      ///< [out] A sequence of (gridDim.x * 2) partial reduction tuples
    Offset                      num_values,             ///< [in] Number of values to reduce
    Offset                      num_segments,           ///< [in] Number of segments being reduced
    Value                       identity,               ///< [in] Identity value (for zero-length segments)
    ReductionOp                 reduction_op,           ///< [in] Reduction operator
    GridEvenShare<Offset>       even_share)             ///< [in] Even-share descriptor for mapping an equal number of tiles onto each thread block
{
    typedef KeyValuePair<Offset, Value> KeyValuePair;

    // Specialize threadblock abstraction type for reducing a region of segmented values
    typedef BlockSegReduceRegion<
            BlockSegReduceRegionPolicy,
            SegmentOffsetIterator,
            ValueIterator,
            OutputIterator,
            ReductionOp,
            Offset>
        BlockSegReduceRegion;

    // Shared memory allocation
    __shared__ typename BlockSegReduceRegion::TempStorage temp_storage;

    // Initialize threadblock even-share to tell us where to start and stop our tile-processing
    even_share.BlockInit();

    // Construct persistent thread block
    BlockSegReduceRegion thread_block(
        temp_storage,
        d_segment_end_offsets,
        d_values,
        d_output,
        num_values,
        num_segments,
        identity,
        reduction_op);

    // First and last partial reduction tuples within the range (valid in thread-0)
    KeyValuePair first_tuple, last_tuple;

    // Consume block's region of work
    thread_block.ProcessBlockRegion(
        even_share.block_offset,
        even_share.block_end,
        first_tuple,
        last_tuple);

    if (threadIdx.x == 0)
    {
        if (gridDim.x > 1)
        {
            // Special case where the first segment written and the carry-out are for the same segment
            if (first_tuple.key == last_tuple.key)
            {
                first_tuple.value = identity;
            }

            // Write the first and last partial products from this thread block so
            // that they can be subsequently "fixed up" in the next kernel.
            d_tuple_partials[blockIdx.x * 2]          = first_tuple;
            d_tuple_partials[(blockIdx.x * 2) + 1]    = last_tuple;
        }
    }

}


/**
 * Segmented reduce region kernel entry point (single-block).
 */
template <
    typename    BlockSegReduceRegionByKeyPolicy,        ///< Parameterized BlockSegReduceRegionByKeyPolicy tuning policy
    typename    InputIterator,                          ///< Random-access iterator referencing key-value input tuples
    typename    OutputIterator,                         ///< Random-access iterator referencing segment output totals
    typename    ReductionOp,                            ///< Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
    typename    Offset,                                 ///< Signed integer type for global offsets
    typename    Value>                                  ///< Value type
__launch_bounds__ (BlockSegReduceRegionByKeyPolicy::BLOCK_THREADS, 1)
__global__ void SegReduceRegionByKeyKernel(
    InputIterator           d_tuple_partials,           ///< [in] A sequence of partial reduction tuples
    OutputIterator          d_output,                   ///< [out] A sequence of \p num_segments segment totals
    Offset                  num_segments,               ///< [in] Number of segments in the \p d_output sequence
    int                     num_tuple_partials,         ///< [in] Number of partial reduction tuples being reduced
    Value                   identity,                   ///< [in] Identity value (for zero-length segments)
    ReductionOp             reduction_op)               ///< [in] Reduction operator
{
    // Specialize threadblock abstraction type for reducing a region of values by key
    typedef BlockSegReduceRegionByKey<
            BlockSegReduceRegionByKeyPolicy,
            InputIterator,
            OutputIterator,
            ReductionOp>
        BlockSegReduceRegionByKey;

    // Shared memory allocation
    __shared__ typename BlockSegReduceRegionByKey::TempStorage temp_storage;

    // Construct persistent thread block
    BlockSegReduceRegionByKey thread_block(
        temp_storage,
        d_tuple_partials,
        d_output,
        identity,
        reduction_op);

    // Process input tiles
    thread_block.ProcessRegion(
        0,                          // Region start
        num_tuple_partials,         // Region end
        0,                          // First segment ID
        num_segments);              // Last segment ID (one-past)
}




/******************************************************************************
 * Dispatch
 ******************************************************************************/

/**
 * Utility class for dispatching the appropriately-tuned kernels for DeviceReduce
 */
template <
    typename ValueIterator,                     ///< Random-access input iterator type for reading values
    typename SegmentOffsetIterator,             ///< Random-access input iterator type for reading segment end-offsets
    typename OutputIterator,                    ///< Random-access output iterator type for writing segment reductions
    typename ReductionOp,                       ///< Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
    typename Offset>                            ///< Signed integer type for global offsets
struct DeviceSegReduceDispatch
{
    // Value type
    typedef typename std::iterator_traits<ValueIterator>::value_type Value;

    // Reduce-by-key data type tuple (segment-ID, value)
    typedef KeyValuePair<Offset, Value> KeyValuePair;



    /******************************************************************************
     * Tuning policies
     ******************************************************************************/

    /// SM35
    struct Policy350
    {
        // ReduceRegionPolicy
        typedef BlockSegReduceRegionPolicy<
                128,                            ///< Threads per thread block
                2, //12,                             ///< Items per thread (per tile of input)
                false,                          ///< Whether or not to cache incoming segment offsets in shared memory before reducing each tile
                false,                          ///< Whether or not to cache incoming values in shared memory before reducing each tile
                LOAD_LDG,                       ///< Cache load modifier for reading segment offsets
                LOAD_LDG,                       ///< Cache load modifier for reading values
                BLOCK_REDUCE_RAKING,            ///< The BlockReduce algorithm to use
                BLOCK_SCAN_RAKING_MEMOIZE>      ///< The BlockScan algorithm to use
            SegReduceRegionPolicy;

        // ReduceRegionByKeyPolicy
        typedef BlockSegReduceRegionByKeyPolicy<
                256,                            ///< Threads per thread block
                9,                             ///< Items per thread (per tile of input)
                BLOCK_LOAD_DIRECT,              ///< The BlockLoad algorithm to use
                false,                          ///< Whether or not only one warp's worth of shared memory should be allocated and time-sliced among block-warps during any load-related data transpositions (versus each warp having its own storage)
                LOAD_LDG,                       ///< Cache load modifier for reading input elements
                BLOCK_SCAN_WARP_SCANS>          ///< The BlockScan algorithm to use
            SegReduceRegionByKeyPolicy;
    };


    /// SM10
    struct Policy100
    {
        // ReduceRegionPolicy
        typedef BlockSegReduceRegionPolicy<
                128,                            ///< Threads per thread block
                3,                              ///< Items per thread (per tile of input)
                false,                          ///< Whether or not to cache incoming segment offsets in shared memory before reducing each tile
                false,                          ///< Whether or not to cache incoming values in shared memory before reducing each tile
                LOAD_DEFAULT,                   ///< Cache load modifier for reading segment offsets
                LOAD_DEFAULT,                   ///< Cache load modifier for reading values
                BLOCK_REDUCE_RAKING,            ///< The BlockReduce algorithm to use
                BLOCK_SCAN_RAKING>              ///< The BlockScan algorithm to use
            SegReduceRegionPolicy;

        // ReduceRegionByKeyPolicy
        typedef BlockSegReduceRegionByKeyPolicy<
                128,                            ///< Threads per thread block
                3,                              ///< Items per thread (per tile of input)
                BLOCK_LOAD_WARP_TRANSPOSE,      ///< The BlockLoad algorithm to use
                false,                          ///< Whether or not only one warp's worth of shared memory should be allocated and time-sliced among block-warps during any load-related data transpositions (versus each warp having its own storage)
                LOAD_DEFAULT,                   ///< Cache load modifier for reading input elements
                BLOCK_SCAN_WARP_SCANS>          ///< The BlockScan algorithm to use
            SegReduceRegionByKeyPolicy;
    };


    /******************************************************************************
     * Tuning policies of current PTX compiler pass
     ******************************************************************************/

#if (CUB_PTX_VERSION >= 350)
    typedef Policy350 PtxPolicy;
/*
#elif (CUB_PTX_VERSION >= 300)
    typedef Policy300 PtxPolicy;

#elif (CUB_PTX_VERSION >= 200)
    typedef Policy200 PtxPolicy;

#elif (CUB_PTX_VERSION >= 130)
    typedef Policy130 PtxPolicy;
*/
#else
    typedef Policy100 PtxPolicy;

#endif

    // "Opaque" policies (whose parameterizations aren't reflected in the type signature)
    struct PtxSegReduceRegionPolicy           : PtxPolicy::SegReduceRegionPolicy {};
    struct PtxSegReduceRegionByKeyPolicy      : PtxPolicy::SegReduceRegionByKeyPolicy {};


    /******************************************************************************
     * Utilities
     ******************************************************************************/

    /**
     * Initialize kernel dispatch configurations with the policies corresponding to the PTX assembly we will use
     */
    template <
        typename SegReduceKernelConfig,
        typename SegReduceByKeyKernelConfig>
    __host__ __device__ __forceinline__
    static void InitConfigs(
        int                         ptx_version,
        SegReduceKernelConfig       &seg_reduce_region_config,
        SegReduceByKeyKernelConfig  &seg_reduce_region_by_key_config)
    {
    #ifdef __CUDA_ARCH__

        // We're on the device, so initialize the kernel dispatch configurations with the current PTX policy
        seg_reduce_region_config.Init<PtxSegReduceRegionPolicy>();
        seg_reduce_region_by_key_config.Init<PtxSegReduceRegionByKeyPolicy>();

    #else

        // We're on the host, so lookup and initialize the kernel dispatch configurations with the policies that match the device's PTX version
        if (ptx_version >= 350)
        {
            seg_reduce_region_config.template          Init<typename Policy350::SegReduceRegionPolicy>();
            seg_reduce_region_by_key_config.template   Init<typename Policy350::SegReduceRegionByKeyPolicy>();
        }
/*
        else if (ptx_version >= 300)
        {
            seg_reduce_region_config.template          Init<typename Policy300::SegReduceRegionPolicy>();
            seg_reduce_region_by_key_config.template   Init<typename Policy300::SegReduceRegionByKeyPolicy>();
        }
        else if (ptx_version >= 200)
        {
            seg_reduce_region_config.template          Init<typename Policy200::SegReduceRegionPolicy>();
            seg_reduce_region_by_key_config.template   Init<typename Policy200::SegReduceRegionByKeyPolicy>();
        }
        else if (ptx_version >= 130)
        {
            seg_reduce_region_config.template          Init<typename Policy130::SegReduceRegionPolicy>();
            seg_reduce_region_by_key_config.template   Init<typename Policy130::SegReduceRegionByKeyPolicy>();
        }
*/
        else
        {
            seg_reduce_region_config.template          Init<typename Policy100::SegReduceRegionPolicy>();
            seg_reduce_region_by_key_config.template   Init<typename Policy100::SegReduceRegionByKeyPolicy>();
        }

    #endif
    }


    /**
     * SegReduceRegionKernel kernel dispatch configuration
     */
    struct SegReduceKernelConfig
    {
        int                     block_threads;
        int                     items_per_thread;
        bool                    use_smem_segment_cache;
        bool                    use_smem_value_cache;
        CacheLoadModifier       load_modifier_segments;
        CacheLoadModifier       load_modifier_values;
        BlockReduceAlgorithm    reduce_algorithm;
        BlockScanAlgorithm      scan_algorithm;

        template <typename SegReduceRegionPolicy>
        __host__ __device__ __forceinline__
        void Init()
        {
            block_threads               = SegReduceRegionPolicy::BLOCK_THREADS;
            items_per_thread            = SegReduceRegionPolicy::ITEMS_PER_THREAD;
            use_smem_segment_cache      = SegReduceRegionPolicy::USE_SMEM_SEGMENT_CACHE;
            use_smem_value_cache        = SegReduceRegionPolicy::USE_SMEM_VALUE_CACHE;
            load_modifier_segments      = SegReduceRegionPolicy::LOAD_MODIFIER_SEGMENTS;
            load_modifier_values        = SegReduceRegionPolicy::LOAD_MODIFIER_VALUES;
            reduce_algorithm            = SegReduceRegionPolicy::REDUCE_ALGORITHM;
            scan_algorithm              = SegReduceRegionPolicy::SCAN_ALGORITHM;
        }
    };

    /**
     * SegReduceRegionByKeyKernel kernel dispatch configuration
     */
    struct SegReduceByKeyKernelConfig
    {
        int                     block_threads;
        int                     items_per_thread;
        BlockLoadAlgorithm      load_algorithm;
        bool                    load_warp_time_slicing;
        CacheLoadModifier       load_modifier;
        BlockScanAlgorithm      scan_algorithm;

        template <typename SegReduceRegionByKeyPolicy>
        __host__ __device__ __forceinline__
        void Init()
        {
            block_threads               = SegReduceRegionByKeyPolicy::BLOCK_THREADS;
            items_per_thread            = SegReduceRegionByKeyPolicy::ITEMS_PER_THREAD;
            load_algorithm              = SegReduceRegionByKeyPolicy::LOAD_ALGORITHM;
            load_warp_time_slicing      = SegReduceRegionByKeyPolicy::LOAD_WARP_TIME_SLICING;
            load_modifier               = SegReduceRegionByKeyPolicy::LOAD_MODIFIER;
            scan_algorithm              = SegReduceRegionByKeyPolicy::SCAN_ALGORITHM;
        }
    };


    /******************************************************************************
     * Dispatch entrypoints
     ******************************************************************************/

    /**
     * Internal dispatch routine for computing a device-wide segmented reduction.
     */
    template <
        typename                        SegReduceRegionKernelPtr,               ///< Function type of hipcub::SegReduceRegionKernel
        typename                        SegReduceRegionByKeyKernelPtr>          ///< Function type of hipcub::SegReduceRegionByKeyKernel
    __host__ __device__ __forceinline__
    static hipError_t Dispatch(
        void                            *d_temp_storage,                        ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
        size_t                          &temp_storage_bytes,                    ///< [in,out] Size in bytes of \p d_temp_storage allocation.
        ValueIterator                   d_values,                               ///< [in] A sequence of \p num_values data to reduce
        SegmentOffsetIterator           d_segment_offsets,                      ///< [in] A sequence of (\p num_segments + 1) segment offsets
        OutputIterator                  d_output,                               ///< [out] A sequence of \p num_segments segment totals
        Offset                          num_values,                             ///< [in] Total number of values to reduce
        Offset                          num_segments,                           ///< [in] Number of segments being reduced
        Value                           identity,                               ///< [in] Identity value (for zero-length segments)
        ReductionOp                     reduction_op,                           ///< [in] Reduction operator
        hipStream_t                    stream,                                 ///< [in] <b>[optional]</b> CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
        bool                            debug_synchronous,                      ///< [in] <b>[optional]</b> Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
        int                             sm_version,                             ///< [in] SM version of target device to use when computing SM occupancy
        SegReduceRegionKernelPtr        seg_reduce_region_kernel,               ///< [in] Kernel function pointer to parameterization of hipcub::SegReduceRegionKernel
        SegReduceRegionByKeyKernelPtr   seg_reduce_region_by_key_kernel,        ///< [in] Kernel function pointer to parameterization of hipcub::SegReduceRegionByKeyKernel
        SegReduceKernelConfig           &seg_reduce_region_config,              ///< [in] Dispatch parameters that match the policy that \p seg_reduce_region_kernel was compiled for
        SegReduceByKeyKernelConfig      &seg_reduce_region_by_key_config)       ///< [in] Dispatch parameters that match the policy that \p seg_reduce_region_by_key_kernel was compiled for
    {
#ifndef CUB_RUNTIME_ENABLED

        // Kernel launch not supported from this device
        return HipcubDebug(hipErrorNotSupported );

#else

        hipError_t error = hipSuccess;
        do
        {
            // Dispatch two kernels: (1) a multi-block segmented reduction
            // to reduce regions by block, and (2) a single-block reduce-by-key kernel
            // to "fix up" segments spanning more than one region.

            // Tile size of seg_reduce_region_kernel
            int tile_size = seg_reduce_region_config.block_threads * seg_reduce_region_config.items_per_thread;

            // Get device ordinal
            int device_ordinal;
            if (HipcubDebug(error = hipGetDevice(&device_ordinal))) break;

            // Get SM count
            int sm_count;
            if (HipcubDebug(error = hipDeviceGetAttribute (&sm_count, hipDeviceAttributeMultiprocessorCount, device_ordinal))) break;

            // Get SM occupancy for histogram_region_kernel
            int seg_reduce_region_sm_occupancy;
            if (HipcubDebug(error = MaxSmOccupancy(
                seg_reduce_region_sm_occupancy,
                sm_version,
                seg_reduce_region_kernel,
                seg_reduce_region_config.block_threads))) break;

            // Get device occupancy for histogram_region_kernel
            int seg_reduce_region_occupancy = seg_reduce_region_sm_occupancy * sm_count;

            // Even-share work distribution
            int num_diagonals = num_values + num_segments;                  // Total number of work items
            int subscription_factor = seg_reduce_region_sm_occupancy;       // Amount of CTAs to oversubscribe the device beyond actively-resident (heuristic)
            GridEvenShare<Offset> even_share(
                num_diagonals,
                seg_reduce_region_occupancy * subscription_factor,
                tile_size);

            // Get grid size for seg_reduce_region_kernel
            int seg_reduce_region_grid_size = even_share.grid_size;

            // Number of "fix-up" reduce-by-key tuples (2 per thread block)
            int num_tuple_partials = seg_reduce_region_grid_size * 2;

            // Temporary storage allocation requirements
            void* allocations[1];
            size_t allocation_sizes[1] =
            {
                num_tuple_partials * sizeof(KeyValuePair),     // bytes needed for "fix-up" reduce-by-key tuples
            };

            // Alias the temporary allocations from the single storage blob (or set the necessary size of the blob)
            if (HipcubDebug(error = AliasTemporaries(d_temp_storage, temp_storage_bytes, allocations, allocation_sizes))) break;
            if (d_temp_storage == NULL)
            {
                // Return if the caller is simply requesting the size of the storage allocation
                return hipSuccess;
            }

            // Alias the allocation for "fix-up" tuples
            KeyValuePair *d_tuple_partials = (KeyValuePair*) allocations[0];

            // Log seg_reduce_region_kernel configuration
            if (debug_synchronous) CubLog("Invoking seg_reduce_region_kernel<<<%d, %d, 0, %lld>>>(), %d items per thread, %d SM occupancy\n",
                seg_reduce_region_grid_size, seg_reduce_region_config.block_threads, (long long) stream, seg_reduce_region_config.items_per_thread, seg_reduce_region_sm_occupancy);

            // Array of segment end-offsets
            SegmentOffsetIterator d_segment_end_offsets = d_segment_offsets + 1;

            // Invoke seg_reduce_region_kernel
            seg_reduce_region_kernel<<<seg_reduce_region_grid_size, seg_reduce_region_config.block_threads, 0, stream>>>(
                d_segment_end_offsets,
                d_values,
                d_output,
                d_tuple_partials,
                num_values,
                num_segments,
                identity,
                reduction_op,
                even_share);

            // Sync the stream if specified
            if (debug_synchronous && (HipcubDebug(error = SyncStream(stream)))) break;

            // Perform "fix-up" of region partial reductions if grid size is greater than one thread block
            if (seg_reduce_region_grid_size > 1)
            {
                // Log seg_reduce_region_by_key_kernel configuration
                if (debug_synchronous) CubLog("Invoking seg_reduce_region_by_key_kernel<<<%d, %d, 0, %lld>>>(), %d items per thread\n",
                    1, seg_reduce_region_by_key_config.block_threads, (long long) stream, seg_reduce_region_by_key_config.items_per_thread);

                // Invoke seg_reduce_region_by_key_kernel
                seg_reduce_region_by_key_kernel<<<1, seg_reduce_region_by_key_config.block_threads, 0, stream>>>(
                    d_tuple_partials,
                    d_output,
                    num_segments,
                    num_tuple_partials,
                    identity,
                    reduction_op);

                // Sync the stream if specified
                if (debug_synchronous && (HipcubDebug(error = SyncStream(stream)))) break;
            }
        }

        while (0);

        return error;

#endif // CUB_RUNTIME_ENABLED
    }


    /**
     * Internal dispatch routine for computing a device-wide segmented reduction.
     */
    __host__ __device__ __forceinline__
    static hipError_t Dispatch(
        void                            *d_temp_storage,                        ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
        size_t                          &temp_storage_bytes,                    ///< [in,out] Size in bytes of \p d_temp_storage allocation.
        ValueIterator                   d_values,                               ///< [in] A sequence of \p num_values data to reduce
        SegmentOffsetIterator           d_segment_offsets,                      ///< [in] A sequence of (\p num_segments + 1) segment offsets
        OutputIterator                  d_output,                               ///< [out] A sequence of \p num_segments segment totals
        Offset                          num_values,                             ///< [in] Total number of values to reduce
        Offset                          num_segments,                           ///< [in] Number of segments being reduced
        Value                           identity,                               ///< [in] Identity value (for zero-length segments)
        ReductionOp                     reduction_op,                           ///< [in] Reduction operator
        hipStream_t                    stream,                                 ///< [in] <b>[optional]</b> CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
        bool                            debug_synchronous)                      ///< [in] <b>[optional]</b> Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
    {
        hipError_t error = hipSuccess;
        do
        {
            // Get PTX version
            int ptx_version;
    #ifndef __CUDA_ARCH__
            if (HipcubDebug(error = PtxVersion(ptx_version))) break;
    #else
            ptx_version = CUB_PTX_VERSION;
    #endif

            // Get kernel kernel dispatch configurations
            SegReduceKernelConfig seg_reduce_region_config;
            SegReduceByKeyKernelConfig seg_reduce_region_by_key_config;

            InitConfigs(ptx_version, seg_reduce_region_config, seg_reduce_region_by_key_config);

            // Dispatch
            if (HipcubDebug(error = Dispatch(
                d_temp_storage,
                temp_storage_bytes,
                d_values,
                d_segment_offsets,
                d_output,
                num_values,
                num_segments,
                identity,
                reduction_op,
                stream,
                debug_synchronous,
                ptx_version,            // Use PTX version instead of SM version because, as a statically known quantity, this improves device-side launch dramatically but at the risk of imprecise occupancy calculation for mismatches
                SegReduceRegionKernel<PtxSegReduceRegionPolicy, SegmentOffsetIterator, ValueIterator, OutputIterator, ReductionOp, Offset, Value>,
                SegReduceRegionByKeyKernel<PtxSegReduceRegionByKeyPolicy, KeyValuePair*, OutputIterator, ReductionOp, Offset, Value>,
                seg_reduce_region_config,
                seg_reduce_region_by_key_config))) break;
        }
        while (0);

        return error;

    }
};




/******************************************************************************
 * DeviceSegReduce
 *****************************************************************************/

/**
 * \brief DeviceSegReduce provides operations for computing a device-wide, parallel segmented reduction across data items residing within global memory.
 * \ingroup DeviceModule
 *
 * \par Overview
 * A <a href="http://en.wikipedia.org/wiki/Reduce_(higher-order_function)"><em>reduction</em></a> (or <em>fold</em>)
 * uses a binary combining operator to compute a single aggregate from a list of input elements.
 *
 * \par Usage Considerations
 * \cdp_class{DeviceReduce}
 *
 */
struct DeviceSegReduce
{
    /**
     * \brief Computes a device-wide segmented reduction using the specified binary \p reduction_op functor.
     *
     * \par
     * Does not support non-commutative reduction operators.
     *
     * \devicestorage
     *
     * \cdp
     *
     * \iterator
     *
     * \tparam ValueIterator            <b>[inferred]</b> Random-access input iterator type for reading values
     * \tparam SegmentOffsetIterator    <b>[inferred]</b> Random-access input iterator type for reading segment end-offsets
     * \tparam OutputIterator           <b>[inferred]</b> Random-access output iterator type for writing segment reductions
     * \tparam Value                    <b>[inferred]</b> Value type
     * \tparam ReductionOp              <b>[inferred]</b> Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
     */
    template <
        typename                ValueIterator,
        typename                SegmentOffsetIterator,
        typename                OutputIterator,
        typename                Value,
        typename                ReductionOp>
    __host__ __device__ __forceinline__
    static hipError_t Reduce(
        void                    *d_temp_storage,                        ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
        size_t                  &temp_storage_bytes,                    ///< [in,out] Size in bytes of \p d_temp_storage allocation.
        ValueIterator           d_values,                               ///< [in] A sequence of \p num_values data to reduce
        SegmentOffsetIterator   d_segment_offsets,                      ///< [in] A sequence of (\p num_segments + 1) segment offsets
        OutputIterator          d_output,                               ///< [out] A sequence of \p num_segments segment totals
        int                     num_values,                             ///< [in] Total number of values to reduce
        int                     num_segments,                           ///< [in] Number of segments being reduced
        Value                   identity,                               ///< [in] Identity value (for zero-length segments)
        ReductionOp             reduction_op,                           ///< [in] Reduction operator
        hipStream_t            stream              = 0,                ///< [in] <b>[optional]</b> CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
        bool                    debug_synchronous   = false)            ///< [in] <b>[optional]</b> Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
    {
        // Signed integer type for global offsets
        typedef int Offset;

        typedef DeviceSegReduceDispatch<
                ValueIterator,
                SegmentOffsetIterator,
                OutputIterator,
                ReductionOp,
                Offset>
            DeviceSegReduceDispatch;

        return DeviceSegReduceDispatch::Dispatch(
            d_temp_storage,
            temp_storage_bytes,
            d_values,
            d_segment_offsets,
            d_output,
            num_values,
            num_segments,
            identity,
            reduction_op,
            stream,
            debug_synchronous);
    }


    /**
     * \brief Computes a device-wide segmented sum using the addition ('+') operator.
     *
     * \par
     * Does not support non-commutative summation.
     *
     * \devicestorage
     *
     * \cdp
     *
     * \iterator
     *
     * \tparam ValueIterator            <b>[inferred]</b> Random-access input iterator type for reading values
     * \tparam SegmentOffsetIterator    <b>[inferred]</b> Random-access input iterator type for reading segment end-offsets
     * \tparam OutputIterator           <b>[inferred]</b> Random-access output iterator type for writing segment reductions
     */
    template <
        typename                ValueIterator,
        typename                SegmentOffsetIterator,
        typename                OutputIterator>
    __host__ __device__ __forceinline__
    static hipError_t Sum(
        void                    *d_temp_storage,                        ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
        size_t                  &temp_storage_bytes,                    ///< [in,out] Size in bytes of \p d_temp_storage allocation.
        ValueIterator           d_values,                               ///< [in] A sequence of \p num_values data to reduce
        SegmentOffsetIterator   d_segment_offsets,                      ///< [in] A sequence of (\p num_segments + 1) segment offsets
        OutputIterator          d_output,                               ///< [out] A sequence of \p num_segments segment totals
        int                     num_values,                             ///< [in] Total number of values to reduce
        int                     num_segments,                           ///< [in] Number of segments being reduced
        hipStream_t            stream              = 0,                ///< [in] <b>[optional]</b> CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
        bool                    debug_synchronous   = false)            ///< [in] <b>[optional]</b> Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
    {
        // Signed integer type for global offsets
        typedef int Offset;

        // Value type
        typedef typename std::iterator_traits<ValueIterator>::value_type Value;

        Value identity = Value();
        hipcub::Sum reduction_op;

        typedef DeviceSegReduceDispatch<
                ValueIterator,
                SegmentOffsetIterator,
                OutputIterator,
                hipcub::Sum,
                Offset>
            DeviceSegReduceDispatch;

        return DeviceSegReduceDispatch::Dispatch(
            d_temp_storage,
            temp_storage_bytes,
            d_values,
            d_segment_offsets,
            d_output,
            num_values,
            num_segments,
            identity,
            reduction_op,
            stream,
            debug_synchronous);
    }
};




//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

/**
 * Initialize problem
 */
template <typename Offset, typename Value>
void Initialize(
    GenMode         gen_mode,
    Value           *h_values,
    vector<Offset>  &segment_offsets,
    int             num_values,
    int             avg_segment_size)
{
    // Initialize values
//    if (g_verbose) printf("Values: ");
    for (int i = 0; i < num_values; ++i)
    {
        InitValue(gen_mode, h_values[i], i);
//        if (g_verbose) std::cout << h_values[i] << ", ";
    }
//    if (g_verbose) printf("\n\n");

    // Initialize segment lengths
    const unsigned int  MAX_INTEGER         = -1u;
    const unsigned int  MAX_SEGMENT_LENGTH  = avg_segment_size * 2;
    const double        SCALE_FACTOR        = double(MAX_SEGMENT_LENGTH) / double(MAX_INTEGER);

    segment_offsets.push_back(0);

    Offset consumed = 0;
    Offset remaining = num_values;
    while (remaining > 0)
    {
        // Randomly sample a 32-bit unsigned int
        unsigned int segment_length;
        RandomBits(segment_length);

        // Scale to maximum segment length
        segment_length = (unsigned int) (double(segment_length) * SCALE_FACTOR);
        segment_length = CUB_MIN(segment_length, remaining);

        consumed += segment_length;
        remaining -= segment_length;

        segment_offsets.push_back(consumed);
    }
}


/**
 * Compute reference answer
 */
template <typename Offset, typename Value>
void ComputeReference(
    Value       *h_values,
    Offset      *h_segment_offsets,
    Value       *h_reference,
    int         num_segments,
    Value       identity)
{
    if (g_verbose) printf("%d segment reductions: ", num_segments);
    for (int segment = 0; segment < num_segments; ++segment)
    {
        h_reference[segment] = identity;

        for (int i = h_segment_offsets[segment]; i < h_segment_offsets[segment + 1]; ++i)
        {
            h_reference[segment] += h_values[i];
        }
        if (g_verbose) std::cout << h_reference[segment] << ", ";
    }
    if (g_verbose) printf("\n\n");
}


/**
 * Simple test of device
 */
template <
    bool            CDP,
    typename        Offset,
    typename        Value,
    typename        ReductionOp>
void Test(
    Offset          num_values,
    int             avg_segment_size,
    ReductionOp     reduction_op,
    Value           identity,
    char*           type_string)
{
    Value   *h_values = NULL;
    Value   *h_reference = NULL;
    Offset  *h_segment_offsets = NULL;

    printf("%d\n", num_values);

    // Initialize problem on host
    h_values = new Value[num_values];
    vector<Offset> segment_offsets;
    Initialize(UNIFORM, h_values, segment_offsets, num_values, avg_segment_size);

    // Allocate simple offsets array and copy STL vector into it
    h_segment_offsets = new Offset[segment_offsets.size()];
    for (int i = 0; i < segment_offsets.size(); ++i)
        h_segment_offsets[i] = segment_offsets[i];

    Offset num_segments = segment_offsets.size() - 1;
    if (g_verbose)
    {
        printf("%d segment offsets: ", num_segments);
        for (int i = 0; i < num_segments; ++i)
            std::cout << h_segment_offsets[i] << "(" << h_segment_offsets[i + 1] - h_segment_offsets[i] << "), ";
        if (g_verbose) std::cout << std::endl << std::endl;
    }

    // Solve problem on host
    h_reference = new Value[num_segments];
    ComputeReference(h_values, h_segment_offsets, h_reference, num_segments, identity);

    printf("\n\n%s hipcub::DeviceSegReduce::%s %d items (%d-byte %s), %d segments (%d-byte offset indices)\n",
        (CDP) ? "CDP device invoked" : "Host-invoked",
        (Equals<ReductionOp, Sum>::VALUE) ? "Sum" : "Reduce",
        num_values, (int) sizeof(Value), type_string,
        num_segments, (int) sizeof(Offset));
    fflush(stdout);

    // Allocate and initialize problem on device
    Value   *d_values = NULL;
    Offset  *d_segment_offsets = NULL;
    Value   *d_output = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values, sizeof(Value) * num_values));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_segment_offsets, sizeof(Offset) * (num_segments + 1)));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_output, sizeof(Value) * num_segments));
    CubDebugExit(hipMemcpy(d_values, h_values, sizeof(Value) * num_values, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_segment_offsets, h_segment_offsets, sizeof(Offset) * (num_segments + 1), hipMemcpyHostToDevice));

    // Request and allocate temporary storage
    void    *d_temp_storage = NULL;
    size_t  temp_storage_bytes = 0;
    CubDebugExit(DeviceSegReduce::Sum(d_temp_storage, temp_storage_bytes, d_values, d_segment_offsets, d_output, num_values, num_segments, 0, false));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Clear device output
    CubDebugExit(hipMemset(d_output, 0, sizeof(Value) * num_segments));

    // Run warmup/correctness iteration
    CubDebugExit(DeviceSegReduce::Sum(d_temp_storage, temp_storage_bytes, d_values, d_segment_offsets, d_output, num_values, num_segments, 0, true));

    // Check for correctness (and display results, if specified)
    int compare = CompareDeviceResults(h_reference, d_output, num_segments, g_verbose, g_verbose);
    printf("\t%s", compare ? "FAIL" : "PASS");

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Performance
    GpuTimer gpu_timer;
    gpu_timer.Start();
    for (int i = 0; i < g_timing_iterations; ++i)
    {
        CubDebugExit(DeviceSegReduce::Sum(d_temp_storage, temp_storage_bytes, d_values, d_segment_offsets, d_output, num_values, num_segments, 0, false));
    }
    gpu_timer.Stop();
    float elapsed_millis = gpu_timer.ElapsedMillis();

    // Display performance
    if (g_timing_iterations > 0)
    {
        float avg_millis = elapsed_millis / g_timing_iterations;
        float grate = float(num_values) / avg_millis / 1000.0 / 1000.0;
        float gbandwidth = grate * sizeof(Value);
        printf(", %.3f avg ms, %.3f billion items/s, %.3f logical GB/s", avg_millis, grate, gbandwidth);
    }

    // Device cleanup
    if (d_values) CubDebugExit(g_allocator.DeviceFree(d_values));
    if (d_segment_offsets) CubDebugExit(g_allocator.DeviceFree(d_segment_offsets));
    if (d_output) CubDebugExit(g_allocator.DeviceFree(d_output));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    // Host cleanup
    if (h_values)           delete[] h_values;
    if (h_segment_offsets)  delete[] h_segment_offsets;
    if (h_reference)        delete[] h_reference;
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_values          = 32 * 1024 * 1024;
    int avg_segment_size    = 500;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_values);
    args.GetCmdLineArgument("ss", avg_segment_size);
    args.GetCmdLineArgument("i", g_timing_iterations);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--v] "
            "[--i=<timing iterations>] "
            "[--n=<input samples>]\n"
            "[--ss=<average segment size>]\n"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    Test<false>((int) num_values, avg_segment_size, Sum(), (double) 0, CUB_TYPE_STRING(double));

    return 0;
}



