/**
 * nvcc invoke_foo.cu -gencode=arch=compute_35,code=\"sm_35,compute_35\" -gencode=arch=compute_20,code=\"sm_20,compute_20\" -Xptxas -v
 * nvcc invoke_foo.cu -gencode=arch=compute_35,code=\"sm_35,compute_35\" -Xptxas -v -rdc=true -lcudadevrt
 */




#include <hip/hip_runtime.h>
#include <stdio.h>

//---------------------------------------------------------------------
// Macro definitions
//---------------------------------------------------------------------

#ifdef __CUDA_ARCH__
#define PTX_ARCH __CUDA_ARCH__
#else
#define PTX_ARCH 0
#endif
#define CNP_ENABLED ((PTX_ARCH == 0) || ((PTX_ARCH >= 350) && defined(__BUILDING_CNPRT__)))


//---------------------------------------------------------------------
// Foo-related kernels and policies
//---------------------------------------------------------------------

/**
 * FooKernel tuning policy
 */
template <int _BLOCK_THREADS, int _ITEMS_PER_THREAD>
struct FooKernelPolicy
{
    enum
    {
        BLOCK_THREADS      = _BLOCK_THREADS,
        ITEMS_PER_THREAD   = _ITEMS_PER_THREAD,
    };
};


/**
 * FooKernel kernel entrypoint
 */
template <typename FooKernelPolicy, typename T>
__launch_bounds__ (FooKernelPolicy::BLOCK_THREADS, 1)
__global__ void FooKernel(T *d_in, T *d_out, int num_elements)
{
    if ((blockIdx.x == 0) && (threadIdx.x == 0)) printf("FooKernel<<<%d, %d>>>, ITEMS_PER_THREAD(%d)\n",
        gridDim.x,
        FooKernelPolicy::BLOCK_THREADS,
        FooKernelPolicy::ITEMS_PER_THREAD);
}


//---------------------------------------------------------------------
// Foo wrapper
//---------------------------------------------------------------------

/**
 * Wrapper for all foo-related entrypoints
 */
struct Foo
{

    /**
     * Invoke foo operation with custom policy
     */
    template <typename FooKernelPolicyT, typename T>
    __host__ __device__ __forceinline__
    static hipError_t Invoke(
        T *d_in,
        T *d_out,
        int num_elements,
        void (*foo_kernel_ptr)(T*, T*, int) = FooKernel<FooKernelPolicyT, T>)
    {
        // Preconfigured tuning policies
        typedef FooKernelPolicy<64,     1>      FooKernelPolicy300;
        typedef FooKernelPolicy<128,    1>      FooKernelPolicy200;
        typedef FooKernelPolicy<256,    1>      FooKernelPolicy100;

        // PTX-specific default policy
    #if PTX_ARCH >= 300
        struct PtxFooKernelPolicy : FooKernelPolicy300 {};
    #elif PTX_ARCH >= 200
        struct PtxFooKernelPolicy : FooKernelPolicy200 {};
    #else
        struct PtxFooKernelPolicy : FooKernelPolicy100 {};
    #endif


        if (foo_kernel_ptr == NULL) foo_kernel_ptr = FooKernel<PtxFooKernelPolicy, T>;

    #if !CNP_ENABLED
        // CUDA API calls and kernel launch not supported from this device
        return cudaErrorInvalidConfiguration;
    #else
        // Determine grid size
        const int TILE_SIZE = FooKernelPolicyT::BLOCK_THREADS * FooKernelPolicyT::ITEMS_PER_THREAD;
        int grid_size = (num_elements + TILE_SIZE - 1) / TILE_SIZE;

        // Invoke kernel
        foo_kernel_ptr<<<grid_size, FooKernelPolicyT::BLOCK_THREADS>>>(d_in, d_out, num_elements);

        return hipSuccess;
    #endif
    }


    /**
     * Invoke foo operation with default policy
     */
    template <typename T>
    __host__ __device__ __forceinline__
    static hipError_t Invoke(T *d_in, T *d_out, int num_elements)
    {
        // Preconfigured tuning policies
        typedef FooKernelPolicy<64,     1>      FooKernelPolicy300;
        typedef FooKernelPolicy<128,    1>      FooKernelPolicy200;
        typedef FooKernelPolicy<256,    1>      FooKernelPolicy100;

        // PTX-specific default policy
    #if PTX_ARCH >= 300
        struct PtxFooKernelPolicy : FooKernelPolicy300 {};
    #elif PTX_ARCH >= 200
        struct PtxFooKernelPolicy : FooKernelPolicy200 {};
    #else
        struct PtxFooKernelPolicy : FooKernelPolicy100 {};
    #endif

    #if !CNP_ENABLED

        // CUDA API calls and kernel launch not supported from this device
        return cudaErrorInvalidConfiguration;

    #else

        // We're on the host, so determine which tuned variant to initialize
        int device_ordinal;
        hipGetDevice(&device_ordinal);

        int major, minor;
        hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device_ordinal);
        hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device_ordinal);
        int device_arch = major * 100 + minor * 10;

        // Our PTX-specific foo kernel function pointer
        void (*foo_kernel_ptr)(T*, T*, int) = FooKernel<PtxFooKernelPolicy, T>;

        // Dispatch with explicit policy
        if (device_arch >= 300)
            return Invoke<FooKernelPolicy300>(d_in, d_out, num_elements, foo_kernel_ptr);
        else if (device_arch >= 200)
            return Invoke<FooKernelPolicy200>(d_in, d_out, num_elements, foo_kernel_ptr);
        else
            return Invoke<FooKernelPolicy100>(d_in, d_out, num_elements, foo_kernel_ptr);

    #endif
    }
};



//---------------------------------------------------------------------
// User kernel for dispatching Foo from device
//---------------------------------------------------------------------

/**
 * User kernel for nested invocation of foo
 */
template <typename T>
__global__ void UserKernel(T *d_in, T *d_out, int num_elements)
{
    // Invoke Foo
    Foo::Invoke(d_in, d_out, num_elements);
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char **argv)
{
    typedef int T;

    T *d_in = NULL;
    T *d_out = NULL;
    int num_elements = 1024 * 1024;

    int dev = 0;
    if (argc > 1)
    {
        dev = atoi(argv[1]);
    }
    hipSetDevice(dev);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, dev);
    printf("Using device %s\n\n", props.name);
    fflush(stdout);

    // Test1: Dispatch Foo from host
    Foo::Invoke(d_in, d_out, num_elements);

    // Test2: Dispatch Foo with custom policy
    Foo::Invoke<FooKernelPolicy<96, 17> >(d_in, d_out, num_elements);

    // Test3: Dispatch user kernel that dispatches Foo from device
    UserKernel<<<1,1>>>(d_in, d_out, num_elements);

    hipDeviceSynchronize();

    return 0;
}
