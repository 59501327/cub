


#include <hip/hip_runtime.h>
#include <stdio.h>



/******************************************************************************
 * FooKernel
 *****************************************************************************/


/// FooKernel tuning policy
template <
    int _BLOCK_THREADS,
    int _ITEMS_PER_THREAD>
struct FooKernelPolicy
{
    enum
    {
        BLOCK_THREADS      = _BLOCK_THREADS,
        ITEMS_PER_THREAD   = _ITEMS_PER_THREAD,
    };
};


/// FooKernel entrypoint
template <
    typename FooKernelPolicy,
    typename T>
__global__ void FooKernel(T *d_in, T *d_out)
{
    if (threadIdx.x == 0) printf("FooKernel BLOCK_THREADS(%d) ITEMS_PER_THREAD(%d)\n",
        FooKernelPolicy::BLOCK_THREADS,
        FooKernelPolicy::ITEMS_PER_THREAD);
}


/******************************************************************************
 * BarKernel
 *****************************************************************************/


/// BarKernel tuning policy
template <
    int _BLOCK_THREADS,
    int _ITEMS_PER_THREAD>
struct BarKernelPolicy
{
    enum
    {
        BLOCK_THREADS      = _BLOCK_THREADS,
        ITEMS_PER_THREAD   = _ITEMS_PER_THREAD,
    };
};


/// BarKernel entrypoint
template <
    typename BarKernelPolicy,
    typename T>
__global__ void BarKernel(T *d_in, T *d_out)
{
    if (threadIdx.x == 0) printf("BarKernel BLOCK_THREADS(%d) ITEMS_PER_THREAD(%d)\n",
        BarKernelPolicy::BLOCK_THREADS,
        BarKernelPolicy::ITEMS_PER_THREAD);
}



/******************************************************************************
 * DeviceBaz
 *****************************************************************************/

#ifdef __CUDA_ARCH__
#define PTX_ARCH __CUDA_ARCH__
#else
#define PTX_ARCH 0
#endif

#define CNP_ENABLED ((PTX_ARCH == 0) || (PTX_ARCH >= 350))


/**
 * Provides BAZ operations on device-global data sets.
 */
struct DeviceBaz
{
    // Configuration context.  These can be configured and then executed later.
    // For example, an autotuning framework may create a large list of
    // different contexts.
    template <typename T>
    struct Context
    {
        //---------------------------------------------------------------------
        // Default tuning types
        //---------------------------------------------------------------------

        typedef FooKernelPolicy<64,     1>      FooKernelPolicy300;
        typedef BarKernelPolicy<64,     1>      BarKernelPolicy300;

        typedef FooKernelPolicy<128,    1>      FooKernelPolicy200;
        typedef BarKernelPolicy<128,    1>      BarKernelPolicy200;

        typedef FooKernelPolicy<256,    1>      FooKernelPolicy100;
        typedef BarKernelPolicy<256,    1>      BarKernelPolicy100;

#if PTX_ARCH >= 300
        struct PtxFooKernelPolicy : FooKernelPolicy300 {};
        struct PtxBarKernelPolicy : BarKernelPolicy300 {};
#elif PTX_ARCH >= 200
        struct PtxFooKernelPolicy : FooKernelPolicy200 {};
        struct PtxBarKernelPolicy : BarKernelPolicy200 {};
#else
        struct PtxFooKernelPolicy : FooKernelPolicy100 {};
        struct PtxBarKernelPolicy : BarKernelPolicy100 {};
#endif

        //---------------------------------------------------------------------
        // Kernel context types
        //---------------------------------------------------------------------

        // Foo kernel context
        struct FooKernelContext
        {
            void (*kernel_ptr)(T *d_in, T *d_out);
            int tile_items;
            int block_threads;

            // Initializer
            template <typename KernelPolicy, typename OpaqueKernelPolicy>
            __host__ __device__ __forceinline__
            hipError_t Init()
            {
                kernel_ptr      = FooKernel<OpaqueKernelPolicy>;
                block_threads   = KernelPolicy::BLOCK_THREADS;
                tile_items      = KernelPolicy::BLOCK_THREADS * KernelPolicy::ITEMS_PER_THREAD;

                return hipSuccess;
            }
        };

        // Bar kernel context
        struct BarKernelContext
        {
            void (*kernel_ptr)(T *d_in, T *d_out);
            int tile_items;
            int block_threads;

            // Initializer
            template <typename KernelPolicy, typename OpaqueKernelPolicy>
            __host__ __device__ __forceinline__ hipError_t
            Init()
            {
                kernel_ptr      = BarKernel<OpaqueKernelPolicy>;
                block_threads   = KernelPolicy::BLOCK_THREADS;
                tile_items      = KernelPolicy::BLOCK_THREADS * KernelPolicy::ITEMS_PER_THREAD;

                return hipSuccess;
            }
        };


        //---------------------------------------------------------------------
        // Fields
        //---------------------------------------------------------------------

        // Foo kernel context
        FooKernelContext foo_kernel_context;

        // Bar kernel context
        BarKernelContext bar_kernel_context;


        //---------------------------------------------------------------------
        // Public interface
        //---------------------------------------------------------------------

        // Initializer (for custom tuning)
        template <typename FooKernelPolicy, typename BarKernelPolicy>
        __host__ __device__ __forceinline__
        hipError_t Init()
        {
            hipError_t retval = hipSuccess;
            do
            {
                if ((retval = foo_kernel_context.template Init<FooKernelPolicy, FooKernelPolicy>())) break;
                if ((retval = bar_kernel_context.template Init<BarKernelPolicy, BarKernelPolicy>())) break;
            }
            while (0);

            return retval;
        }


        // Initializer (for default tuning)
        __host__ __device__ __forceinline__
        hipError_t Init()
        {
            hipError_t retval = hipSuccess;
            do
            {

#if PTX_ARCH > 0
                if ((retval = foo_kernel_context.template Init<PtxFooKernelPolicy, PtxFooKernelPolicy>())) break;
                if ((retval = bar_kernel_context.template Init<PtxBarKernelPolicy, PtxBarKernelPolicy>())) break;
#else
                int gpu_ordinal;
                hipGetDevice(&gpu_ordinal);
                hipDeviceProp_t device_props;
                hipGetDeviceProperties(&device_props, gpu_ordinal);
                int device_arch = device_props.major * 100 + device_props.minor * 10;

                if (device_arch >= 300)
                {
                    if ((retval = foo_kernel_context.template Init<FooKernelPolicy300, PtxFooKernelPolicy>())) break;
                    if ((retval = bar_kernel_context.template Init<BarKernelPolicy300, PtxBarKernelPolicy>())) break;
                }
                else if (device_arch >= 200)
                {
                    if ((retval = foo_kernel_context.template Init<FooKernelPolicy200, PtxFooKernelPolicy>())) break;
                    if ((retval = bar_kernel_context.template Init<BarKernelPolicy200, PtxBarKernelPolicy>())) break;
                }
                else
                {
                    if ((retval = foo_kernel_context.template Init<FooKernelPolicy100, PtxFooKernelPolicy>())) break;
                    if ((retval = bar_kernel_context.template Init<BarKernelPolicy100, PtxBarKernelPolicy>())) break;
                }
#endif
            }
            while (0);

            return retval;
        }


        // Invoke operation (instance must be initialized first)
        __host__ __device__ __forceinline__
        hipError_t Baz(T *d_in, T *d_out)
        {
            hipError_t retval = hipSuccess;

        #if CNP_ENABLED
            do
            {
                int grid_size = 1;
                foo_kernel_context.kernel_ptr<<<grid_size, foo_kernel_context.block_threads>>>(d_in, d_out);
                bar_kernel_context.kernel_ptr<<<grid_size, bar_kernel_context.block_threads>>>(d_in, d_out);
            }
            while (0);
        #endif

            return retval;
        }

    };

    // Baz
    template <typename T>
    __host__ __device__ __forceinline__
    static hipError_t Baz(T *d_in, T *d_out)
    {
        hipError_t retval = hipSuccess;
        do
        {
            // Create configuration instance
            Context<T> config_instance;
            if ((retval = config_instance.Init())) break;

            // Run
            if ((retval = config_instance.Baz(d_in, d_out))) break;
        }
        while (0);

        return retval;
    }

};


/******************************************************************************
 * Main
 *****************************************************************************/

template <typename T>
__global__ void WrapperKernel(T *d_in, T *d_out)
{
    if (threadIdx.x == 0) printf("WrapperKernel\n");

#if CNP_ENABLED
    if ((blockIdx.x == 0) && (threadIdx.x == 0))
    {
        // Cuda nested kernel invocation
        DeviceBaz::Baz(d_in, d_out);
    }

    int gpu_ordinal;
    hipGetDevice(&gpu_ordinal);

    int sm_count;
    hipDeviceGetAttribute (&sm_count, hipDeviceAttributeMultiprocessorCount, gpu_ordinal);
    printf("Sm count: %d\n", sm_count);

#endif
}


/// Main
int main()
{
    typedef int T;

    T *d_in = NULL;
    T *d_out = NULL;

    DeviceBaz::Baz(d_in, d_out);

    WrapperKernel<<<1,1>>>(d_in, d_out);

    hipDeviceSynchronize();

    return 0;
}



