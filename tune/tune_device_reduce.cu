#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of DeviceReduce utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <vector>
#include <algorithm>
#include <stdio.h>
#include <cub.cuh>
#include "../test/test_util.h"

using namespace hipcub;
using namespace std;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

#ifndef TUNE_ARCH
#define TUNE_ARCH 100
#endif

int     g_max_items         = 48 * 1024 * 1024;
int     g_samples           = 100;
int     g_iterations        = 2;
int     g_min_exponent      = 14;       // min sampled problem size is 2^14 (16384)
bool    g_verbose           = false;
bool    g_single            = false;


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize problem
 */
template <typename T>
void Initialize(
    int             gen_mode,
    T               *h_in,
    int             num_items)
{
    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
    }
}

/**
 * Sequential reduction
 */
template <typename T, typename ReductionOp>
T Reduce(
    T               *h_in,
    ReductionOp     reduction_op,
    int             num_items)
{
    T retval = h_in[0];
    for (int i = 1; i < num_items; ++i)
        retval = reduction_op(retval, h_in[i]);
    return retval;
}



//---------------------------------------------------------------------
// Full tile test generation
//---------------------------------------------------------------------



/**
 * Wrapper structure for generating and running different tuning configurations
 */
template <
    typename T,
    typename SizeT,
    typename ReductionOp>
struct Schmoo
{
    //---------------------------------------------------------------------
    // Types
    //---------------------------------------------------------------------

    /// Pairing of kernel function pointer and corresponding dispatch params
    template <typename KernelPtr>
    struct DispatchTuple
    {
        KernelPtr                           kernel_ptr;
        DeviceReduce::KernelDispachParams   params;

        float                               avg_throughput;
        float                               max_throughput;
        int                                 cumulative_rank;

        DispatchTuple() : kernel_ptr(0), avg_throughput(0.0), max_throughput(0.0), cumulative_rank(0) {}
    };

    /**
     * Comparison operator for DispatchTuple.avg_throughput
     */
    template <typename Tuple>
    static bool MinThroughput(const Tuple &a, const Tuple &b)
    {
        return (a.avg_throughput < b.avg_throughput);
    }

    /**
     * Comparison operator for DispatchTuple.cumulative_rank
     */
    template <typename Tuple>
    static bool MinRank(const Tuple &a, const Tuple &b)
    {
        return (a.cumulative_rank < b.cumulative_rank);
    }


    /// Multi-block reduction kernel type and dispatch tuple type
    typedef void (*MultiReduceKernelPtr)(T*, T*, SizeT, GridEvenShare<SizeT>, GridQueue<SizeT>, ReductionOp);
    typedef DispatchTuple<MultiReduceKernelPtr> MultiDispatchTuple;

    /// Single-block reduction kernel type and dispatch tuple type
    typedef void (*SingleReduceKernelPtr)(T*, T*, SizeT, ReductionOp);
    typedef DispatchTuple<SingleReduceKernelPtr> SingleDispatchTuple;


    //---------------------------------------------------------------------
    // Fields
    //---------------------------------------------------------------------

    vector<MultiDispatchTuple> multi_kernels;       // List of generated multi-block kernels
    vector<SingleDispatchTuple> single_kernels;     // List of generated single-block kernels


    //---------------------------------------------------------------------
    // Kernel enumeration methods
    //---------------------------------------------------------------------

    /**
     * Specialization that allows kernel generation with the specified BlockReduceTilesPolicy
     */
    template <
        typename BlockReduceTilesPolicy,
        bool IsOk = (sizeof(typename BlockReduceTiles<BlockReduceTilesPolicy, T*, SizeT>::SmemStorage) < ArchProps<TUNE_ARCH>::SMEM_BYTES)>
    struct Ok
    {
        /// Enumerate multi-block kernel and add to the list
        template <typename KernelsVector>
        static void GenerateMulti(
            KernelsVector &multi_kernels,
            int subscription_factor)
        {
            MultiDispatchTuple tuple;
            tuple.params.template Init<BlockReduceTilesPolicy>(subscription_factor);
            tuple.kernel_ptr = MultiReduceKernel<BlockReduceTilesPolicy, T*, T*, SizeT, ReductionOp>;
            multi_kernels.push_back(tuple);
        }


        /// Enumerate single-block kernel and add to the list
        template <typename KernelsVector>
        static void GenerateSingle(KernelsVector &single_kernels)
        {
            SingleDispatchTuple tuple;
            tuple.params.template Init<BlockReduceTilesPolicy>();
            tuple.kernel_ptr = SingleReduceKernel<BlockReduceTilesPolicy, T*, T*, SizeT, ReductionOp>;
            single_kernels.push_back(tuple);
        }
    };

    /**
     * Specialization that rejects kernel generation with the specified BlockReduceTilesPolicy
     */
    template <typename BlockReduceTilesPolicy>
    struct Ok<BlockReduceTilesPolicy, false>
    {
        template <typename KernelsVector>
        static void GenerateMulti(KernelsVector &multi_kernels) {}

        template <typename KernelsVector>
        static void GenerateSingle(KernelsVector &single_kernels) {}
    };


    /// Enumerate block-scheduling variations
    template <
        int             BLOCK_THREADS,
        int             ITEMS_PER_THREAD,
        int             VECTOR_LOAD_LENGTH,
        PtxLoadModifier LOAD_MODIFIER>
    void Enumerate()
    {
        // Multi-block kernels
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, LOAD_MODIFIER, GRID_MAPPING_EVEN_SHARE> >::GenerateMulti(multi_kernels, 1);
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, LOAD_MODIFIER, GRID_MAPPING_EVEN_SHARE> >::GenerateMulti(multi_kernels, 2);
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, LOAD_MODIFIER, GRID_MAPPING_EVEN_SHARE> >::GenerateMulti(multi_kernels, 4);
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, LOAD_MODIFIER, GRID_MAPPING_EVEN_SHARE> >::GenerateMulti(multi_kernels, 8);
#if TUNE_ARCH >= 200
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, LOAD_MODIFIER, GRID_MAPPING_DYNAMIC> >::GenerateMulti(multi_kernels, 1);
#endif

        // Single-block kernels
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, LOAD_MODIFIER, GRID_MAPPING_EVEN_SHARE> >::GenerateSingle(single_kernels);
    }


    /// Enumerate load modifier variations
    template <
        int BLOCK_THREADS,
        int ITEMS_PER_THREAD,
        int VECTOR_LOAD_LENGTH>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, PTX_LOAD_NONE>();
#if TUNE_ARCH >= 350
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, PTX_LOAD_LDG>();
#endif
    }


    /// Enumerate vectorization variations
    template <
        int BLOCK_THREADS,
        int ITEMS_PER_THREAD>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, 1>();
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, 2>();
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, 4>();
    }


    /// Enumerate thread-granularity variations
    template <int BLOCK_THREADS>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, 1>();
        Enumerate<BLOCK_THREADS, 2>();
        Enumerate<BLOCK_THREADS, 4>();
//      Enumerate<BLOCK_THREADS, 7>();
        Enumerate<BLOCK_THREADS, 8>();
//      Enumerate<BLOCK_THREADS, 11>();
//      Enumerate<BLOCK_THREADS, 12>();
//      Enumerate<BLOCK_THREADS, 15>();
        Enumerate<BLOCK_THREADS, 16>();
//      Enumerate<BLOCK_THREADS, 19>();
        Enumerate<BLOCK_THREADS, 20>();
    }


    /// Enumerate block size variations
    void Enumerate()
    {
        printf("\nEnumerating kernels\n"); fflush(stdout);

        Enumerate<32>();
        Enumerate<64>();
        Enumerate<96>();
        Enumerate<128>();
        Enumerate<160>();
        Enumerate<192>();
        Enumerate<256>();
        Enumerate<512>();
    }


    //---------------------------------------------------------------------
    // Multi-block test methods
    //---------------------------------------------------------------------

    /**
     * Test multi reduction
     */
    void Test(
        MultiDispatchTuple      &multi_dispatch,
        SingleDispatchTuple     &single_dispatch,
        T*                      d_in,
        T*                      d_out,
        T*                      h_reference,
        SizeT                   num_items,
        ReductionOp             reduction_op)
    {
        // Clear output
        CubDebugExit(hipMemset(d_out, 0, sizeof(T)));

        // Warmup/correctness iteration
        DeviceReduce::Dispatch(
            multi_dispatch.kernel_ptr,
            single_dispatch.kernel_ptr,
            multi_dispatch.params,
            single_dispatch.params,
            d_in,
            d_out,
            num_items,
            reduction_op);

        CubDebugExit(hipDeviceSynchronize());

        // Copy out and display results
        int correct = CompareDeviceResults(h_reference, d_out, 1, true, false);

        // Performance
        GpuTimer gpu_timer;
        float elapsed_millis = 0.0;
        for (int i = 0; i < g_iterations; i++)
        {
            gpu_timer.Start();

            DeviceReduce::Dispatch(
                multi_dispatch.kernel_ptr,
                single_dispatch.kernel_ptr,
                multi_dispatch.params,
                single_dispatch.params,
                d_in,
                d_out,
                num_items,
                reduction_op);

            gpu_timer.Stop();
            elapsed_millis += gpu_timer.ElapsedMillis();
        }

        float avg_elapsed = elapsed_millis / g_iterations;
        float avg_throughput = float(num_items) / avg_elapsed / 1000.0 / 1000.0;
        float avg_bandwidth = avg_throughput * sizeof(T);

        multi_dispatch.avg_throughput = CUB_MAX(avg_throughput, multi_dispatch.avg_throughput);
        multi_dispatch.max_throughput = CUB_MAX(avg_throughput, multi_dispatch.max_throughput);

        single_dispatch.avg_throughput = CUB_MAX(avg_throughput, single_dispatch.avg_throughput);
        single_dispatch.max_throughput = CUB_MAX(avg_throughput, single_dispatch.max_throughput);

        if (g_verbose)
        {
            printf("\t%.2f GB/s, multi_dispatch( ", avg_bandwidth);
            multi_dispatch.params.Print();
            printf(" ), single_dispatch( ");
            single_dispatch.params.Print();
            printf(" )\n");
            fflush(stdout);
        }

        AssertEquals(0, correct);
    }


    /**
     * Evaluate multi-block configurations
     */
    void TestMulti(
        T*                      h_in,
        T*                      d_in,
        T*                      d_out,
        ReductionOp             reduction_op)
    {
        // Simple single kernel tuple for use with multi kernel sweep
        typedef typename DeviceReduce::TunedPolicies<T, SizeT, TUNE_ARCH>::SinglePolicy SimpleSinglePolicy;
        SingleDispatchTuple simple_single_tuple;
        simple_single_tuple.params.template Init<SimpleSinglePolicy>();
        simple_single_tuple.kernel_ptr = SingleReduceKernel<SimpleSinglePolicy, T*, T*, SizeT, ReductionOp>;

        double max_exponent      = log2(double(g_max_items));
        unsigned int max_int     = (unsigned int) -1;

        for (int sample = 0; sample < g_samples; ++sample)
        {
            printf("\nMulti-block sample %d,", sample);

            int num_items;
            if (sample == 0)
            {
                // First sample: use max items
                num_items = g_max_items;
                printf("num_items: %d", num_items); fflush(stdout);
            }
            else
            {
                // Sample a problem size from [2^g_min_exponent, g_max_items].  First 2/3 of the samples are log-distributed, the other 1/3 are uniformly-distributed.
                unsigned int bits;
                RandomBits(bits);
                double scale = double(bits) / max_int;

                if (sample < (2 * g_samples) / 3)
                {
                    // log bias
                    double exponent = ((max_exponent - g_min_exponent) * scale) + g_min_exponent;
                    num_items = pow(2.0, exponent);
                    printf("num_items: %d (2^%.2f)", num_items, exponent); fflush(stdout);
                }
                else
                {
                    // uniform bias
                    num_items = CUB_MAX(pow(2.0, g_min_exponent), scale * g_max_items);
                    printf("num_items: %d (%.2f * %d)", num_items, scale, g_max_items); fflush(stdout);
                }
            }
            if (g_verbose)
                printf("\n");
            else
                printf(", ");

            // Compute reference
            T h_reference = Reduce(h_in, reduction_op, num_items);

            // Run test on each multi-kernel configuration
            for (int j = 0; j < multi_kernels.size(); ++j)
            {
                multi_kernels[j].avg_throughput = 0.0;
                Test(multi_kernels[j], simple_single_tuple, d_in, d_out, &h_reference, num_items, reduction_op);
            }

            // Sort by throughput
            sort(multi_kernels.begin(), multi_kernels.end(), MinThroughput<MultiDispatchTuple>);

            // Print best throughput for this problem size
            float best_throughput = (multi_kernels.size() > 0) ? multi_kernels.back().avg_throughput : 0;
            printf("Best: %.2fe9 items/s (%.2f GB/s)\n", best_throughput, best_throughput * sizeof(T));

            // Update cumulative rank
            for (int j = 0; j < multi_kernels.size(); ++j)
                multi_kernels[j].cumulative_rank += j;
        }

        // Sort by cumulative rank
        sort(multi_kernels.begin(), multi_kernels.end(), MinRank<MultiDispatchTuple>);

        // Find max overall throughput
        float overall_max_throughput = 0.0;
        for (int j = 0; j < multi_kernels.size(); ++j)
            overall_max_throughput = CUB_MAX(overall_max_throughput, multi_kernels[j].max_throughput);

        // Print ranked multi configurations
        printf("\nRanked multi_kernels:\n");
        for (int j = 0; j < multi_kernels.size(); ++j)
        {
            printf("\t (%d) params( ", multi_kernels.size() - j);
            multi_kernels[j].params.Print();
            printf(" ) avg rank: %.2f, max throughput %.2f (%.2f GB/s, %.2f%%)\n",
                float(multi_kernels[j].cumulative_rank) / (g_samples * multi_kernels.size()),
                multi_kernels[j].max_throughput,
                multi_kernels[j].max_throughput * sizeof(T),
                multi_kernels[j].max_throughput / overall_max_throughput);
        }

        printf("\nMax multi-block throughput %.2f (%.2f GB/s)\n", overall_max_throughput, overall_max_throughput * sizeof(T));
    }


    //---------------------------------------------------------------------
    // Single-block test methods
    //---------------------------------------------------------------------

    /**
     * Test single reduction
     */
    void Test(
        SingleDispatchTuple     &single_dispatch,
        T*                      d_in,
        T*                      d_out,
        T*                      h_reference,
        SizeT                   num_items,
        ReductionOp             reduction_op)
    {
        // Clear output
        CubDebugExit(hipMemset(d_out, 0, sizeof(T)));

        // Warmup/correctness iteration
        DeviceReduce::DispatchSingle(
            single_dispatch.kernel_ptr,
            single_dispatch.params,
            d_in,
            d_out,
            num_items,
            reduction_op);

        CubDebugExit(hipDeviceSynchronize());

        // Copy out and display results
        int correct = CompareDeviceResults(h_reference, d_out, 1, true, false);

        // Performance
        GpuTimer gpu_timer;
        float elapsed_millis = 0.0;
        for (int i = 0; i < g_iterations; i++)
        {
            gpu_timer.Start();

            DeviceReduce::DispatchSingle(
                single_dispatch.kernel_ptr,
                single_dispatch.params,
                d_in,
                d_out,
                num_items,
                reduction_op);

            gpu_timer.Stop();
            elapsed_millis += gpu_timer.ElapsedMillis();
        }

        float avg_elapsed = elapsed_millis / g_iterations;
        float avg_throughput = float(num_items) / avg_elapsed / 1000.0 / 1000.0;
        float avg_bandwidth = avg_throughput * sizeof(T);

        single_dispatch.avg_throughput = CUB_MAX(avg_throughput, single_dispatch.avg_throughput);
        single_dispatch.max_throughput = CUB_MAX(avg_throughput, single_dispatch.max_throughput);

        if (g_verbose)
        {
            printf("\t%.2f GB/s, single_dispatch( ", avg_bandwidth);
            single_dispatch.params.Print();
            printf(" )\n");
            fflush(stdout);
        }

        AssertEquals(0, correct);
    }


    /**
     * Evaluate single-block configurations
     */
    void TestSingle(
        T*                      h_in,
        T*                      d_in,
        T*                      d_out,
        ReductionOp             reduction_op)
     {
        double max_exponent     = log2(double(g_max_items));
        unsigned int max_int    = (unsigned int) -1;

        for (int sample = 0; sample < g_samples; ++sample)
        {
            printf("\nSingle-block sample %d,", sample);

            int num_items;
            if (sample == 0)
            {
                // First sample: use max items
                num_items = g_max_items;
                printf("num_items: %d", num_items); fflush(stdout);
            }
            else
            {
                // Sample a problem size from [2, g_max_items], log-distributed
                unsigned int bits;
                RandomBits(bits);
                double scale = double(bits) / max_int;
                double exponent = ((max_exponent - 1) * scale) + 1;
                num_items = pow(2.0, exponent);
                printf("num_items: %d (2^%.2f)", num_items, exponent); fflush(stdout);
            }

            if (g_verbose)
                printf("\n");
            else
                printf(", ");

            // Compute reference
            T h_reference = Reduce(h_in, reduction_op, num_items);

            // Run test on each single-kernel configuration
            for (int j = 0; j < single_kernels.size(); ++j)
            {
                single_kernels[j].avg_throughput = 0.0;
                Test(single_kernels[j], d_in, d_out, &h_reference, num_items, reduction_op);
            }

            // Sort by throughput
            sort(single_kernels.begin(), single_kernels.end(), MinThroughput<SingleDispatchTuple>);

            // Print best throughput for this problem size
            float best_throughput = (single_kernels.size() > 0) ? single_kernels.back().avg_throughput : 0;
            printf("Best: %.2fe9 items/s (%.2f GB/s)\n", best_throughput, best_throughput * sizeof(T));

            // Update cumulative rank
            for (int j = 0; j < single_kernels.size(); ++j)
                single_kernels[j].cumulative_rank += j;
        }

        // Sort by cumulative rank
        sort(single_kernels.begin(), single_kernels.end(), MinRank<SingleDispatchTuple>);

        // Find max overall throughput
        float overall_max_throughput = 0.0;
        for (int j = 0; j < single_kernels.size(); ++j)
            overall_max_throughput = CUB_MAX(overall_max_throughput, single_kernels[j].max_throughput);

        // Print ranked single configurations
        printf("\nRanked single_kernels:\n");
        for (int j = 0; j < single_kernels.size(); ++j)
        {
            printf("\t (%d) params( ", single_kernels.size() - j);
            single_kernels[j].params.Print();
            printf(" ) avg rank: %.2f, max throughput %.2f (%.2f GB/s, %.2f%%)\n",
                float(single_kernels[j].cumulative_rank) / (g_samples * single_kernels.size()),
                single_kernels[j].max_throughput,
                single_kernels[j].max_throughput * sizeof(T),
                single_kernels[j].max_throughput / overall_max_throughput);
        }

        printf("\nMax single-block throughput %.2f (%.2f GB/s)\n", overall_max_throughput, overall_max_throughput * sizeof(T));
    }

};



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    args.GetCmdLineArgument("n", g_max_items);
    args.GetCmdLineArgument("s", g_samples);
    args.GetCmdLineArgument("i", g_iterations);
    g_verbose = args.CheckCmdLineFlag("v");
    g_single = args.CheckCmdLineFlag("single");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--n=<max items>]"
            "[--s=<samples>]"
            "[--i=<timing iterations>]"
            "[--v]"
            "[--single]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    typedef unsigned int SizeT;
    typedef unsigned int T;
    Sum<T> reduction_op;

    // Enumerate kernels
    Schmoo<T, SizeT, Sum<T> > schmoo;
    schmoo.Enumerate();

    // Allocate host arrays
    T *h_in = new T[g_max_items];

    // Initialize problem
    Initialize(UNIFORM, h_in, g_max_items);

    // Initialize device arrays
    T *d_in = NULL;
    T *d_out = NULL;
    CubDebugExit(DeviceAllocate((void**)&d_in, sizeof(T) * g_max_items));
    CubDebugExit(DeviceAllocate((void**)&d_out, sizeof(T) * 1));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * g_max_items, hipMemcpyHostToDevice));

    // Test kernels
    if (g_single)
        schmoo.TestSingle(h_in, d_in, d_out, reduction_op);
    else
        schmoo.TestMulti(h_in, d_in, d_out, reduction_op);

    // Cleanup
    if (h_in) delete[] h_in;
    if (d_in) CubDebugExit(DeviceFree(d_in));
    if (d_out) CubDebugExit(DeviceFree(d_out));

    return 0;
}



